#include "hip/hip_runtime.h"
#include "FractalClass.cuh"
#include <iostream>
#include <thread>
#include <functional>

bool running_other_core = false;

sf::Image stretchImageNearestNeighbor(const sf::Image& source, unsigned int targetWidth, unsigned int targetHeight) {
    sf::Image result({ targetWidth, targetHeight }, sf::Color::Black);

    float scaleX = static_cast<float>(source.getSize().x) / targetWidth;
    float scaleY = static_cast<float>(source.getSize().y) / targetHeight;

    for (unsigned int y = 0; y < targetHeight; ++y) {
        for (unsigned int x = 0; x < targetWidth; ++x) {
            unsigned int srcX = static_cast<unsigned int>(x * scaleX);
            unsigned int srcY = static_cast<unsigned int>(y * scaleY);
            sf::Color color = source.getPixel({ srcX, srcY });
            result.setPixel({ x, y }, color);
        }
    }

    return result;
}

template <typename Derived>
FractalBase<Derived>::FractalBase()
    : max_iterations(300), basic_zoom_x(240.0f), basic_zoom_y(240.0f),
    zoom_x(basic_zoom_x), zoom_y(basic_zoom_y),
    x_offset(3.5f), y_offset(2.5f),
    zoom_factor(1.0f), zoom_speed(0.1f),
    pixels(new unsigned char[width * height * 4]), paletteSize(palette.size()),
    zoom_scale(1.0f), width(400), height(300)
{
    hipMalloc(&stopFlagDevice, sizeof(bool));
    bool flag = true;
    hipMemcpy(stopFlagDevice, &flag, sizeof(bool), hipMemcpyHostToDevice);

    hipMalloc(&d_palette, palette.size() * sizeof(sf::Color));
    hipMemcpy(d_palette, palette.data(), palette.size() * sizeof(sf::Color), hipMemcpyHostToDevice);

    hipMallocManaged(&d_pixels, width * height * sizeof(uint32_t));
}

template <typename Derived>
FractalBase<Derived>::~FractalBase() {
    hipFree(d_palette);
    hipFree(d_pixels);
    hipFree(stopFlagDevice);
    delete[] pixels;
}

template <typename Derived>
unsigned int FractalBase<Derived>::get_max_iters() { return max_iterations; }

template <typename Derived>
bool FractalBase<Derived>::get_is_dragging() { return is_dragging; }

template <typename Derived>
void FractalBase<Derived>::set_max_iters(unsigned int max_iters) { max_iterations = max_iters; }

template <typename Derived>
double FractalBase<Derived>::get_x_offset() { return x_offset; }

template <typename Derived>
double FractalBase<Derived>::get_y_offset() { return y_offset; }

template <typename Derived>
double FractalBase<Derived>::get_zoom_x() { return zoom_x; }

template <typename Derived>
double FractalBase<Derived>::get_zoom_y() { return zoom_y; }

template <typename Derived>
double FractalBase<Derived>::get_zoom_scale() { return zoom_scale; }

template <typename Derived>
void FractalBase<Derived>::checkEventAndSetFlag(hipEvent_t event) {
    while (hipEventQuery(event) == hipErrorNotReady) {
        std::this_thread::sleep_for(std::chrono::milliseconds(1));
    }
    bool flag = false;
    running_other_core = false;
    hipMemcpy(stopFlagDevice, &flag, sizeof(bool), hipMemcpyHostToDevice);
}

void FractalBase<fractals::mandelbrot>::render(render_state quality) {
    hipEvent_t event;
    hipEventCreate(&event);

    int new_width, new_height;
    double new_zoom_scale;

    if (quality == render_state::good) {
        new_width = 800;
        new_height = 600;
        antialiasing = false;
        new_zoom_scale = 1.0;
    }
    else { // render_state::best
        new_width = 1600;
        new_height = 1200;
        antialiasing = true;
        new_zoom_scale = 2.0;
    }

    if (width != new_width || height != new_height) {
        double center_x = x_offset + (width / (zoom_x * zoom_scale)) / 2.0;
        double center_y = y_offset + (height / (zoom_y * zoom_scale)) / 2.0;
        
        zoom_scale = new_zoom_scale;
        width = new_width;
        height = new_height;

        x_offset = center_x - (width / (zoom_x * zoom_scale)) / 2.0;
        y_offset = center_y - (height / (zoom_y * zoom_scale)) / 2.0;

        delete[] pixels;
        pixels = new unsigned char[new_width * new_height * 4];

        bool flag = true;
        hipMemcpy(stopFlagDevice, &flag, sizeof(bool), hipMemcpyHostToDevice);
        hipDeviceSynchronize();

        hipFree(d_pixels);
        hipError_t allocErr = hipMalloc(&d_pixels, new_width * new_height * sizeof(uint32_t));
        if (allocErr != hipSuccess) {
            std::cerr << "CUDA Memory Allocation Error: " << hipGetErrorString(allocErr) << std::endl;
            exit(EXIT_FAILURE);
        }

        width = new_width;
        height = new_height;
    }

    double render_zoom_x = zoom_x * zoom_scale;
    double render_zoom_y = zoom_y * zoom_scale;

    dim3 dimBlock(64, 64);
    dim3 dimGrid(
        (width + dimBlock.x - 1) / dimBlock.x,
        (height + dimBlock.y - 1) / dimBlock.y
    );
    if(running_other_core)
        hipDeviceSynchronize();
    running_other_core = true;
    fractal_rendering <<<dimBlock, dimGrid>>> (
        d_pixels, width, height, render_zoom_x, render_zoom_y,
        x_offset, y_offset, d_palette, paletteSize,
        max_iterations, stopFlagDevice
        );

    hipEventRecord(event);

    std::thread eventChecker([this, event]() {
        hipEventSynchronize(event);
        bool flag = false;
        hipMemcpy(stopFlagDevice, &flag, sizeof(bool), hipMemcpyHostToDevice);
        });

    eventChecker.join();
    hipEventDestroy(event);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << hipGetErrorString(err);
    }
}


void FractalBase<fractals::julia>::render(
    render_state quality,
    double mandel_x_offset, double mandel_y_offset,
    double mandel_zoom_x, double mandel_zoom_y,
    double cx, double cy
) {
    hipEvent_t event;
    hipEventCreate(&event);

    int new_width, new_height;
    double new_zoom_scale;

    if (quality == render_state::good) {
        new_width = 800;
        new_height = 600;
        antialiasing = false;
        new_zoom_scale = 1.0;
    }
    else {
        new_width = 1600;
        new_height = 1200;
        antialiasing = true;
        new_zoom_scale = 2.0;
    }

    if (width != new_width || height != new_height) {
        double center_x = x_offset + (width / (zoom_x * zoom_scale)) / 2.0;
        double center_y = y_offset + (height / (zoom_y * zoom_scale)) / 2.0;

        zoom_scale = new_zoom_scale;
        width = new_width;
        height = new_height;

        x_offset = center_x - (width / (zoom_x * zoom_scale)) / 2.0;
        y_offset = center_y - (height / (zoom_y * zoom_scale)) / 2.0;

        delete[] pixels;
        pixels = new unsigned char[new_width * new_height * 4];

        bool flag = true;
        hipMemcpy(stopFlagDevice, &flag, sizeof(bool), hipMemcpyHostToDevice);
        hipDeviceSynchronize();

        hipFree(d_pixels);
        hipError_t allocErr = hipMalloc(&d_pixels, new_width * new_height * sizeof(uint32_t));
        if (allocErr != hipSuccess) {
            std::cerr << "CUDA Memory Allocation Error: " << hipGetErrorString(allocErr) << std::endl;
            exit(EXIT_FAILURE);
        }

        width = new_width;
        height = new_height;
    }

    double render_zoom_x = zoom_x * zoom_scale;
    double render_zoom_y = zoom_y * zoom_scale;

    dim3 dimBlock(64, 64);
    dim3 dimGrid(
        (width + dimBlock.x - 1) / dimBlock.x,
        (height + dimBlock.y - 1) / dimBlock.y
    );
    if (running_other_core)
        hipDeviceSynchronize();
    running_other_core = true;

    fractal_rendering << <dimBlock, dimGrid >> > (
        d_pixels, width, height, render_zoom_x, render_zoom_y,
        x_offset, y_offset, d_palette, paletteSize,
        max_iterations, stopFlagDevice, cx, cy
        );

    hipEventRecord(event);

    std::thread eventChecker([this, event]() {
        hipEventSynchronize(event);
        bool flag = false;
        hipMemcpy(stopFlagDevice, &flag, sizeof(bool), hipMemcpyHostToDevice);
        });

    eventChecker.join();
    hipEventDestroy(event);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << hipGetErrorString(err) << std::endl;
    }
}


template <typename Derived>
void FractalBase<Derived>::draw(sf::RenderTarget& target, sf::RenderStates states) const {
    hipMemcpy(pixels, d_pixels, width * height * 4 * sizeof(unsigned char), hipMemcpyDeviceToHost);
    sf::Image img1({ width, height }, pixels);
    sf::Image image;
    if (antialiasing) {
        unsigned char* pix_dest;
		hipMalloc(&pix_dest, 800 * 600 * 4 * sizeof(unsigned char));

        dim3 dimBlock(64, 64);
        dim3 dimGrid(
            (width + dimBlock.x - 1) / dimBlock.x,
            (height + dimBlock.y - 1) / dimBlock.y);
        if(running_other_core)
			hipDeviceSynchronize();
        ANTIALIASING_SSAA4<<<dimBlock, dimGrid>>>(d_pixels, pix_dest, 1600, 1200, 800, 600);
        hipDeviceSynchronize();

        unsigned char* compressed = new unsigned char[800 * 600 * 4];
		hipMemcpy(compressed, pix_dest, 800 * 600 * 4 * sizeof(unsigned char), hipMemcpyDeviceToHost);

        image = sf::Image({ 800, 600 }, compressed);
        hipFree(pix_dest);
    }
    else {
        image = img1;
    }

    sf::Texture texture;
    texture.loadFromImage(image);
    sf::Sprite sprite(texture);

    sprite.setPosition({ 0, 0 });

    states.transform *= getTransform();
    target.draw(sprite, states);
}
template <typename Derived>
void FractalBase<Derived>::handleZoom(float wheel_delta, const sf::Vector2i mouse_pos) {
    double old_zoom_x = zoom_x;
    double old_zoom_y = zoom_y;
    double old_x_offset = x_offset;
    double old_y_offset = y_offset;

    double zoom_change = 1.0 + wheel_delta * zoom_speed;
    zoom_factor *= zoom_change;
    zoom_factor = std::max(std::min(zoom_factor, 100000000000000.0), 0.01);

    zoom_x = basic_zoom_x * zoom_factor;
    zoom_y = basic_zoom_y * zoom_factor;

    double image_mouse_x = mouse_pos.x * 1.0;
    double image_mouse_y = mouse_pos.y * 1.0;
    if (std::is_same<Derived, fractals::julia>::value) {
        image_mouse_x -= 1920 - 800;
    }

    x_offset = old_x_offset + (image_mouse_x / zoom_x - image_mouse_x / old_zoom_x);
    y_offset = old_y_offset + (image_mouse_y / zoom_y - image_mouse_y / old_zoom_y);

    std::cout << "Zoom Factor: " << zoom_factor << ", Offset: (" << x_offset << ", " << y_offset << ")" << std::endl;
}

template <typename Derived>
void FractalBase<Derived>::start_dragging(sf::Vector2i mouse_pos) {
    is_dragging = true;
    drag_start_pos = mouse_pos;
}


template <typename Derived>
void FractalBase<Derived>::dragging(sf::Vector2i mouse_pos) {
    if (!is_dragging) return;

    sf::Vector2i delta_pos = mouse_pos - drag_start_pos;
    double delta_real = static_cast<double>(delta_pos.x) / (zoom_x * zoom_scale);
    double delta_imag = static_cast<double>(delta_pos.y) / (zoom_y * zoom_scale);

    x_offset += delta_real;
    y_offset += delta_imag;
    drag_start_pos = mouse_pos;
}

template <typename Derived>
void FractalBase<Derived>::stop_dragging() {
    is_dragging = false;
}

template class FractalBase<fractals::mandelbrot>;
template class FractalBase<fractals::julia>;