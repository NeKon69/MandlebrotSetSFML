#include "hip/hip_runtime.h"
#include "FractalClass.cuh"
#include <iostream>
#include <thread>
#include <functional>

bool running_other_core = false;

sf::Image stretchImageNearestNeighbor(const sf::Image& source, unsigned int targetWidth, unsigned int targetHeight) {
    sf::Image result({ targetWidth, targetHeight }, sf::Color::Black);

    float scaleX = static_cast<float>(source.getSize().x) / targetWidth;
    float scaleY = static_cast<float>(source.getSize().y) / targetHeight;

    for (unsigned int y = 0; y < targetHeight; ++y) {
        for (unsigned int x = 0; x < targetWidth; ++x) {
            unsigned int srcX = static_cast<unsigned int>(x * scaleX);
            unsigned int srcY = static_cast<unsigned int>(y * scaleY);
            sf::Color color = source.getPixel({ srcX, srcY });
            result.setPixel({ x, y }, color);
        }
    }

    return result;
}

template <typename Derived>
FractalBase<Derived>::FractalBase()
    : max_iterations(300), basic_zoom_x(240.0), basic_zoom_y(240.0),
    zoom_x(basic_zoom_x), zoom_y(basic_zoom_y),
    x_offset(3.0), y_offset(1.825),
    zoom_factor(1.0), zoom_speed(0.1),
    pixels(new unsigned char[width * height * 4]), paletteSize(palette.size()),
    zoom_scale(1.0), width(400), height(300)
{
    if (std::is_same<Derived, fractals::julia>::value) {
        x_offset = 1.8;
        y_offset = 1.25;
    }
    hipMalloc(&stopFlagDevice, sizeof(bool));
    bool flag = true;
    hipMemcpy(stopFlagDevice, &flag, sizeof(bool), hipMemcpyHostToDevice);
    stopFlagCpu.store(flag);

    hipMalloc(&d_palette, palette.size() * sizeof(sf::Color));
    hipMemcpy(d_palette, palette.data(), palette.size() * sizeof(sf::Color), hipMemcpyHostToDevice);

    // Alloc data for the GPU uncompressed image
    hipMallocManaged(&d_pixels, 1600 * 1200 * 4 * sizeof(uint32_t));

    // Alloc data for the CPU uncompressed image
    hipHostMalloc(&pixels, 1600 * 1200 * 4 * sizeof(char4));

    // Alloc data for the GPU compressed image
    hipMalloc(&ssaa_buffer, 800 * 600 * 4 * sizeof(char4));

    // Alloc data for the CPU compressed image
	hipHostMalloc(&compressed, 800 * 600 * 4 * sizeof(char4));
}

template <typename Derived>
FractalBase<Derived>::~FractalBase() {
    hipFree(d_palette);
    hipFree(d_pixels);
    hipFree(stopFlagDevice);
    hipHostFree(pixels);
}

template <typename Derived>
unsigned int FractalBase<Derived>::get_max_iters() { return max_iterations; }

template <typename Derived>
bool FractalBase<Derived>::get_is_dragging() { return is_dragging; }

template <typename Derived>
void FractalBase<Derived>::set_max_iters(unsigned int max_iters) { max_iterations = max_iters; }

template <typename Derived>
double FractalBase<Derived>::get_x_offset() { return x_offset; }

template <typename Derived>
double FractalBase<Derived>::get_y_offset() { return y_offset; }

template <typename Derived>
double FractalBase<Derived>::get_zoom_x() { return zoom_x; }

template <typename Derived>
double FractalBase<Derived>::get_zoom_y() { return zoom_y; }

template <typename Derived>
double FractalBase<Derived>::get_zoom_scale() { return zoom_scale; }


// that code served me good in the past, however it's being replaced with better version with atomic operations
// UwU sooooo saaaad UwU
//template <typename Derived>
//void FractalBase<Derived>::checkEventAndSetFlag(hipEvent_t event) {
//    while (hipEventQuery(event) == hipErrorNotReady) {
//        std::this_thread::sleep_for(std::chrono::milliseconds(1));
//    }
//    bool flag = false;
//    running_other_core = false;
//    hipMemcpy(stopFlagDevice, &flag, sizeof(bool), hipMemcpyHostToDevice);
//}

void FractalBase<fractals::mandelbrot>::render(render_state quality) {
    hipEvent_t event;
    hipEventCreate(&event);

    int new_width, new_height;
    double new_zoom_scale;

    if (quality == render_state::good) {
        new_width = 800;
        new_height = 600;
        antialiasing = false;
        new_zoom_scale = 1.0;
    }
    else { // render_state::best
        new_width = 1600;
        new_height = 1200;
        antialiasing = true;
        new_zoom_scale = 2.0;
    }

    if (width != new_width || height != new_height) {
        double center_x = x_offset + (width / (zoom_x * zoom_scale)) / 2.0;
        double center_y = y_offset + (height / (zoom_y * zoom_scale)) / 2.0;

        zoom_scale = new_zoom_scale;
        width = new_width;
        height = new_height;

        x_offset = center_x - (width / (zoom_x * zoom_scale)) / 2.0;
        y_offset = center_y - (height / (zoom_y * zoom_scale)) / 2.0;

        bool flag = true;
        hipMemcpy(stopFlagDevice, &flag, sizeof(bool), hipMemcpyHostToDevice);
        stopFlagCpu.store(flag);

        hipDeviceSynchronize();

        width = new_width;
        height = new_height;
    }

    double render_zoom_x = zoom_x * zoom_scale;
    double render_zoom_y = zoom_y * zoom_scale;

    dim3 dimBlock(64, 64);
    dim3 dimGrid(
        (width + dimBlock.x - 1) / dimBlock.x,
        (height + dimBlock.y - 1) / dimBlock.y
    );
    if (running_other_core)
        hipDeviceSynchronize();
    running_other_core = true;

	size_t len = width * height * 4;
    fractal_rendering <<<dimBlock, dimGrid>>> (
        d_pixels, len, width, height, render_zoom_x, render_zoom_y,
        x_offset, y_offset, d_palette, paletteSize,
        max_iterations, stopFlagDevice
        );
    stopFlagCpu.store(false);

    hipEventRecord(event);

    std::thread eventChecker([this, event]() {
        while (hipGetLastError() == hipErrorNotReady) {
            std::this_thread::sleep_for(std::chrono::milliseconds(1));
        }
        bool flag = false;
        running_other_core = false;
        hipMemcpy(stopFlagDevice, &flag, sizeof(bool), hipMemcpyHostToDevice);
        hipEventDestroy(event);

        stopFlagCpu.store(false);
        });

    eventChecker.detach();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << "fractal mandelbrot: " << hipGetErrorString(err) << "\n";
    }
}


void FractalBase<fractals::julia>::render(
    render_state quality,
    double zx, double zy
) {
    hipEvent_t event;
    hipEventCreate(&event);

    int new_width, new_height;
    double new_zoom_scale;

    if (quality == render_state::good) {
        new_width = 800;
        new_height = 600;
        antialiasing = false;
        new_zoom_scale = 1.0;
    }
    else { // render_state::best
        new_width = 1600;
        new_height = 1200;
        antialiasing = true;
        new_zoom_scale = 2.0;
    }

    if (width != new_width || height != new_height) {
        double center_x = x_offset + (width / (zoom_x * zoom_scale)) / 2.0;
        double center_y = y_offset + (height / (zoom_y * zoom_scale)) / 2.0;

        zoom_scale = new_zoom_scale;
        width = new_width;
        height = new_height;

        x_offset = center_x - (width / (zoom_x * zoom_scale)) / 2.0;
        y_offset = center_y - (height / (zoom_y * zoom_scale)) / 2.0;

        bool flag = true;
        hipMemcpy(stopFlagDevice, &flag, sizeof(bool), hipMemcpyHostToDevice);
        stopFlagCpu.store(flag);

        hipDeviceSynchronize();

        width = new_width;
        height = new_height;
    }

    double render_zoom_x = zoom_x * zoom_scale;
    double render_zoom_y = zoom_y * zoom_scale;

    dim3 dimBlock(64, 64);
    dim3 dimGrid(
        (width + dimBlock.x - 1) / dimBlock.x,
        (height + dimBlock.y - 1) / dimBlock.y
    );
    if (running_other_core)
        hipDeviceSynchronize();
    running_other_core = true;

    size_t len = width * height * 4;

    fractal_rendering<<<dimBlock, dimGrid>>>(
        d_pixels, len, width, height, render_zoom_x, render_zoom_y,
        x_offset, y_offset, d_palette, paletteSize,
        max_iterations, stopFlagDevice, zx, zy
        );
    stopFlagCpu.store(false);

    hipEventRecord(event);

    std::thread eventChecker([this, event]() {
        while(hipGetLastError() == hipErrorNotReady) {
			std::this_thread::sleep_for(std::chrono::milliseconds(1));
		}
        bool flag = false;
        running_other_core = false;
        hipMemcpy(stopFlagDevice, &flag, sizeof(bool), hipMemcpyHostToDevice);
        hipEventDestroy(event);

        stopFlagCpu.store(false);
        });

    eventChecker.detach();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << "fractal julia: " << hipGetErrorString(err) << "\n";
    }
}


template <typename Derived>
void FractalBase<Derived>::draw(sf::RenderTarget& target, sf::RenderStates states) const {
    if (antialiasing) {
        // SSAA rendering
        dim3 dimBlock(50, 50);
        dim3 dimGrid(
            (width + dimBlock.x - 1) / dimBlock.x,
            (height + dimBlock.y - 1) / dimBlock.y
        );
        if (running_other_core) hipDeviceSynchronize();
        running_other_core = false;
        auto start = std::chrono::high_resolution_clock::now();
        ANTIALIASING_SSAA4<<<dimBlock, dimGrid>>>(d_pixels, ssaa_buffer, 1600, 1200, 800, 600);
        hipDeviceSynchronize();
		auto end = std::chrono::high_resolution_clock::now();
		std::cout << "SSAA4 time: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() << " ms" << std::endl;

        auto start_copying = std::chrono::high_resolution_clock::now();
        hipMemcpy(compressed, ssaa_buffer, 800 * 600 * 4 * sizeof(unsigned char), hipMemcpyDeviceToHost);

        sf::Image image({ 800, 600 }, compressed);
        sf::Texture texture(image);
        sf::Sprite sprite(texture);

		auto end_copying = std::chrono::high_resolution_clock::now();
		std::cout << "SSAA4 copying time: " << std::chrono::duration_cast<std::chrono::milliseconds>(end_copying - start_copying).count() << " ms" << std::endl;

        sprite.setPosition({ 0, 0 });
        states.transform *= getTransform();
        target.draw(sprite, states);
    }
    else {
        hipMemcpy(pixels, d_pixels, width * height * 4 * sizeof(unsigned char), hipMemcpyDeviceToHost);

        sf::Image image({ 800, 600 }, pixels);
        sf::Texture texture(image);
        sf::Sprite sprite(texture);

        sprite.setPosition({ 0, 0 });
        states.transform *= getTransform();
        target.draw(sprite, states);
    }
}
template <typename Derived>
void FractalBase<Derived>::handleZoom(float wheel_delta, const sf::Vector2i mouse_pos) {
    double old_zoom_x = zoom_x;
    double old_zoom_y = zoom_y;
    double old_x_offset = x_offset;
    double old_y_offset = y_offset;

    double zoom_change = 1.0 + wheel_delta * zoom_speed;
    zoom_factor *= zoom_change;
    zoom_factor = std::max(std::min(zoom_factor, 100000000000000.0), 0.01);

    zoom_x = basic_zoom_x * zoom_factor;
    zoom_y = basic_zoom_y * zoom_factor;

    double image_mouse_x = mouse_pos.x * 1.0;
    double image_mouse_y = mouse_pos.y * 1.0;
    if (std::is_same<Derived, fractals::julia>::value) {
        image_mouse_x -= 1920 - 800;
    }

    x_offset = old_x_offset + (image_mouse_x / zoom_x - image_mouse_x / old_zoom_x);
    y_offset = old_y_offset + (image_mouse_y / zoom_y - image_mouse_y / old_zoom_y);

}

template <typename Derived>
void FractalBase<Derived>::start_dragging(sf::Vector2i mouse_pos) {
    is_dragging = true;
    drag_start_pos = mouse_pos;
}


template <typename Derived>
void FractalBase<Derived>::dragging(sf::Vector2i mouse_pos) {
    if (!is_dragging) return;

    sf::Vector2i delta_pos = mouse_pos - drag_start_pos;
    double delta_real = static_cast<double>(delta_pos.x) / (zoom_x * zoom_scale);
    double delta_imag = static_cast<double>(delta_pos.y) / (zoom_y * zoom_scale);

    x_offset += delta_real;
    y_offset += delta_imag;
    drag_start_pos = mouse_pos;
}

template <typename Derived>
void FractalBase<Derived>::stop_dragging() {
    is_dragging = false;
}

template class FractalBase<fractals::mandelbrot>;
template class FractalBase<fractals::julia>;