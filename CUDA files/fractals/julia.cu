#include "hip/hip_runtime.h"
#include "julia.cuh"
#include <iostream>

__global__ void fractal_rendering(
	unsigned char* pixels, size_t size_of_pixels, int width, int height,
	float zoom_x, float zoom_y, float x_offset, float y_offset,
	sf::Color* d_palette, int paletteSize, float maxIterations, bool* stopFlagDevice,
	float cReal, float cImaginary) {

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	size_t expected_size = width * height * 4;

	float scale_factor = (float)size_of_pixels / expected_size;



	if (x < width && y < height) {
		float real = cReal;
		float imag = cImaginary;
		float new_real, new_imag;
		float z_real = x / zoom_x - x_offset;;
		float z_imag = y / zoom_y - y_offset;
		float current_iteration = 0;

		while (complex_abs2(z_real, z_imag) < 4 && current_iteration < maxIterations) {
			new_real = complex_mult_real(z_real, z_imag, z_real, z_imag) + real;
			new_imag = complex_mult_imag(z_real, z_imag, z_real, z_imag) + imag;
			z_real = new_real;
			z_imag = new_imag;
			current_iteration++;

		}

		unsigned char r, g, b;
		if (current_iteration == maxIterations) {
			r = g = b = 0; 
		}

		else {
			// Smooth iteration count
			current_iteration = current_iteration + 1 - dev_log2(dev_log2(dev_abs(dev_sqrt(complex_abs2(z_real, z_imag)))));
			// Calculate gradient value
			float gradient = Gradient(current_iteration, maxIterations);
			// Map gradient to palette index
			int index = static_cast<int>(gradient * (paletteSize - 1));
			sf::Color color = getPaletteColor(index, paletteSize, d_palette);
			r = color.r;
			g = color.g;
			b = color.b;
		}

		int base_index = (y * width + x) * 4;
		for (int i = 0; i < scale_factor * 4; i += 4) {
			int index = base_index + i;
			pixels[index] = r;
			pixels[index + 1] = g;
			pixels[index + 2] = b;
			pixels[index + 3] = 255;
		}
	}

}
