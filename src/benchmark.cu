#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include "benchmark.cuh"

__global__ void benchmarkKernel(float* out, const float* a, const float* b, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        float sum = 0.0f;
        for (int i = 0; i < 100000; ++i) {
            sum += a[idx] * b[idx] + a[idx] - b[idx];
        }
        out[idx] = sum;
    }
}

float measureGFLOPS(int N) {
    float* d_a, * d_b, * d_out;
    hipMalloc(&d_a, N * sizeof(float));
    hipMalloc(&d_b, N * sizeof(float));
    hipMalloc(&d_out, N * sizeof(float));

    float* h_a = new float[N];
    float* h_b = new float[N];
    for (int i = 0; i < N; i++) {
        h_a[i] = 1.0f;
        h_b[i] = 2.0f;
    }
    hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);
    delete[] h_a;
    delete[] h_b;

    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    benchmarkKernel << <gridSize, blockSize >> > (d_out, d_a, d_b, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    float total_flops = N * 100000.0f * 4;
    float gflops = (total_flops / (milliseconds * 1e6));

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    return gflops;
}