#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include "benchmark.cuh"

__global__ void benchmarkKernel(float* out, const float* a, const float* b, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        float sum = 0.0f;
        for (int i = 0; i < 100000; ++i) {
            sum += a[idx] * b[idx] + a[idx] - b[idx];
        }
        out[idx] = sum;
    }
}

__global__ void benchmarkKernel(double* out, const double* a, const double* b, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        double sum = 0.0;
        for (int i = 0; i < 100000; ++i) {
            sum += a[idx] * b[idx] + a[idx] - b[idx];
        }
        out[idx] = sum;
    }
}

float measureGFLOPS(int N) {
    float* d_a, * d_b, * d_out;
    hipMalloc(&d_a, N * sizeof(float));
    hipMalloc(&d_b, N * sizeof(float));
    hipMalloc(&d_out, N * sizeof(float));

    float* h_a = new float[N];
    float* h_b = new float[N];
    for (int i = 0; i < N; i++) {
        h_a[i] = 1.0f;
        h_b[i] = 2.0f;
    }
    hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);
    delete[] h_a;
    delete[] h_b;

    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    benchmarkKernel << <gridSize, blockSize >> > (d_out, d_a, d_b, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    float total_flops = N * 100000.0f * 4;
    float gflops = (total_flops / (milliseconds * 1e6));

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    return gflops;
}

double measureGDFLOPS(int N) {
    double *d_a, *d_b, *d_out;
    hipMalloc(&d_a, N * sizeof(double));
    hipMalloc(&d_b, N * sizeof(double));
    hipMalloc(&d_out, N * sizeof(double));
    double* h_a = new double[N];
    double* h_b = new double[N];
    for (int i = 0; i < N; i++) {
        h_a[i] = 1.0;
        h_b[i] = 2.0;
    }
    hipMemcpy(d_a, h_a, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(double), hipMemcpyHostToDevice);
    delete[] h_a;
    delete[] h_b;
    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    benchmarkKernel<<<gridSize, blockSize>>>(d_out, d_a, d_b, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    double total_Dflops = N * 100000.0 * 4;
    double gDflops = (total_Dflops / (milliseconds * 1e6));
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    return gDflops;
}