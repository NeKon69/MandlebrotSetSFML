#include "ClassImplementation/Fractals/JuliaRendering.cuh"
#include "ClassImplementation/Fractals/MandelbrotRendering.cuh"
#include "ClassImplementation/CustomFormulaHandling.h"
#include "ClassImplementation/Processing.cu"
#include "ClassImplementation/JuliaTimelapse.h"
#include "ClassImplementation/FractalInteraction.h"
#include "ClassImplementation/PaletteHandler.h"
#include "ClassImplementation/IterationPath.h"
#include <TGUI/Backend/SFML-Graphics.hpp>

template <typename Derived>
FractalBase<Derived>::FractalBase()
    :
    thread_stop_flags(std::thread::hardware_concurrency() * 100),
    max_iterations(MAX_ITERATIONS), basic_zoom_x(BASIC_ZOOM_X), basic_zoom_y(BASIC_ZOOM_Y),
    zoom_x(basic_zoom_x), zoom_y(basic_zoom_y),
    x_offset(BASIC_X_OFFSET), y_offset(BASIC_Y_OFFSET),
    zoom_factor(BASIC_ZOOM_FACTOR), zoom_speed(BASIC_ZOOM_SPEED),
    zoom_scale(BASIC_ZOOM_SCALE),  maxComputationF(BASIC_MAX_COMPUTATION_F), maxComputationD(BASIC_MAX_COMPUTATION_D),
    basic_width(BASIC_WIDTH), basic_height(BASIC_HEIGHT),
    width(basic_width), height(basic_height),
    sprite(texture), iterationline(sf::PrimitiveType::LineStrip),
    gen(rd()),
    disX(-2.f, 2.f), disY(-1.5f, 1.5f),
    disVelX(-0.13f, 0.13f),disVelY(-0.1f, 0.1f)
{
    initialized_nvrtc = false;
    created_context = false;

    isCudaAvailable = true;
    int numDevices = 0;
    hipGetDeviceCount(&numDevices);
    if(numDevices == 0) {
        std::cout << "IMPORTANT NO AVAILABLE CUDA DEVICES FOUND" << std::endl;
        std::cout << "Forcing to use CPU rendering" << std::endl;
        std::cout << "Please make sure you have CUDA installed and your GPU supports it" << std::endl;
        isCudaAvailable = false;
    }
    else {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, 0);
        compute_capability = "--gpu-architecture=compute_" + std::to_string(deviceProp.major) + std::to_string(deviceProp.minor);
    }
    palette = createHSVPalette(20000);
    paletteSize = 20000;


    ALLOCATE_ALL_IMAGE_MEMORY();
    ALLOCATE_ALL_NON_IMAGE_MEMORY();

    iterationpoints.resize(max_iterations);

}

template <typename Derived>
FractalBase<Derived>::~FractalBase() {
    FREE_ALL_IMAGE_MEMORY();
    FREE_ALL_NON_IMAGE_MEMORY();
    if (module_loaded) CU_SAFE_CALL(hipModuleUnload(module));
    MAKE_CURR_CONTEXT_OPERATION(hipFree(nullptr), hipCtxDestroy(ctx), context);
}

template <typename Derived>
unsigned int FractalBase<Derived>::get_max_iters() { return max_iterations; }

template <typename Derived>
bool FractalBase<Derived>::get_is_dragging() { return is_dragging; }

template <typename Derived>
void FractalBase<Derived>::set_max_iters(unsigned int max_iters) { max_iterations = max_iters; }

template <typename Derived>
double FractalBase<Derived>::get_x_offset() { return x_offset; }

template <typename Derived>
double FractalBase<Derived>::get_y_offset() { return y_offset; }

template <typename Derived>
double FractalBase<Derived>::get_zoom_x() { return zoom_x; }

template <typename Derived>
double FractalBase<Derived>::get_zoom_y() { return zoom_y; }

template <typename Derived>
double FractalBase<Derived>::get_zoom_scale() { return zoom_scale; }

template <typename Derived>
double FractalBase<Derived>::get_hardness_coeff() { return hardness_coeff; }

template <typename Derived>
sf::Texture FractalBase<Derived>::getTexture() { return texture; }

template <typename Derived>
void FractalBase<Derived>::setMaxComputation(float Gflops, float GDflops) { maxComputationF = 50.0f / 90 * Gflops; maxComputationD = 50.0f / 90 * GDflops; }

template <typename Derived>
Palletes FractalBase<Derived>::getPallete() { return curr_pallete; }

template <typename Derived>
sf::Vector2i FractalBase<Derived>::get_resolution() const { return {int(basic_width), int(basic_height)}; }

template <typename Derived>
unsigned int FractalBase<Derived>::get_compiling_percentage() {
    if (context == context_type::NVRTC) {
        return progress_compiling_percentage;
    }
    else {
        return 0;
    }
}

template <typename Derived>
void FractalBase<Derived>::set_resolution(sf::Vector2i target_resolution) {
    unsigned int old_width = width, old_height = height;

    width = basic_width = target_resolution.x;
    height = basic_height = target_resolution.y;

    double center_x = x_offset + (old_width / (zoom_x * zoom_scale)) / 2.0;
    double center_y = y_offset + (old_height / (zoom_y * zoom_scale)) / 2.0;

    zoom_x = basic_zoom_x * zoom_factor;
    zoom_y = basic_zoom_y * zoom_factor;

    x_offset = center_x - (width / (zoom_x * zoom_scale)) / 2.0;
    y_offset = center_y - (height / (zoom_y * zoom_scale)) / 2.0;

    FREE_ALL_IMAGE_MEMORY();
    ALLOCATE_ALL_IMAGE_MEMORY();
}


// To surely not forget anything lets make sure to delete everything and reallocate
template <typename Derived>
void FractalBase<Derived>::reset() {
    // Free existing resources
    FREE_ALL_IMAGE_MEMORY();
    FREE_ALL_NON_IMAGE_MEMORY();

    INIT_BASIC_VALUES;

    if (std::is_same<Derived, fractals::julia>::value) {
        x_offset = 2.5;
        palette = createHSVPalette(BASIC_PALETTE_SIZE);
        paletteSize = BASIC_PALETTE_SIZE;
    }
    else {
        palette = createHSVPalette(BASIC_PALETTE_SIZE);
        paletteSize = BASIC_PALETTE_SIZE;
    }

    if(context == context_type::NVRTC) hipCtxSetCurrent(ctx);
    ALLOCATE_ALL_IMAGE_MEMORY();
    ALLOCATE_ALL_NON_IMAGE_MEMORY();

    iterationpoints.resize(max_iterations);
}

// that code served me good in the past, however it's being replaced with better version with atomic operations
// nevermind, atomic sucks!!! async is the way to go
// UwU sooooo saaaad UwU
//template <typename Derived>
//void FractalBase<Derived>::checkEventAndSetFlag(hipEvent_t event) {
//    while (hipEventQuery(event) == hipErrorNotReady) {
//        std::this_thread::sleep_for(std::chrono::milliseconds(1));
//    }
//    bool flag = false;
//    running_other_core = false;
//    hipMemcpy(stopFlagDevice, &flag, sizeof(bool), hipMemcpyHostToDevice);
//}

template <typename Derived>
context_type FractalBase<Derived>::get_context() { return context; }

template <typename Derived>
bool FractalBase<Derived>::get_bool_custom_formula() { return custom_formula; }

template class FractalBase<fractals::mandelbrot>;
template class FractalBase<fractals::julia>;