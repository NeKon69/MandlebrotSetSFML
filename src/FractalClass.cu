/// Includes necessary headers for fractal rendering (Julia, Mandelbrot),
/// custom formula handling (NVRTC), CUDA processing utilities,
/// Julia set timelapse functionality, fractal interaction logic (zoom, pan),
/// color palette management, iteration path drawing, and TGUI backend integration.
#include "ClassImplementation/Fractals/JuliaRendering.cuh"
#include "ClassImplementation/Fractals/MandelbrotRendering.cuh"
#include "ClassImplementation/CustomFormulaHandling.h"
#include "ClassImplementation/Processing.cu"
#include "ClassImplementation/JuliaTimelapse.h"
#include "ClassImplementation/FractalInteraction.h"
#include "ClassImplementation/PaletteHandler.h"
#include "ClassImplementation/IterationPath.h"
#include <TGUI/Backend/SFML-Graphics.hpp>

/// Template class FractalBase implementation.
/// Provides common functionality for different fractal types (Mandelbrot, Julia).
/// Manages fractal parameters, rendering state, memory, and user interaction.
template <typename Derived>
FractalBase<Derived>::FractalBase()
        :
        /// Initializes flags for potential thread management
        thread_stop_flags(std::thread::hardware_concurrency() * 100),
        /// Initializes basic and current fractal parameters like iterations, zoom, offset, and speed.
        max_iterations(MAX_ITERATIONS), basic_zoom_x(BASIC_ZOOM_X), basic_zoom_y(BASIC_ZOOM_Y),
        zoom_x(basic_zoom_x), zoom_y(basic_zoom_y),
        x_offset(BASIC_X_OFFSET), y_offset(BASIC_Y_OFFSET),
        zoom_factor(BASIC_ZOOM_FACTOR), zoom_speed(BASIC_ZOOM_SPEED),
        zoom_scale(BASIC_ZOOM_SCALE),  maxComputationF(BASIC_MAX_COMPUTATION_F), maxComputationD(BASIC_MAX_COMPUTATION_D),
        /// Initializes basic and current resolution.
        basic_width(BASIC_WIDTH), basic_height(BASIC_HEIGHT),
        width(basic_width), height(basic_height),
        /// Initializes SFML sprite and texture for rendering, and primitive type for iteration lines.
        sprite(texture), iterationline(sf::PrimitiveType::LineStrip),
        /// Sets up random number generators for timelapse.
        gen(rd()),
        disX(-2.f, 2.f), disY(-1.5f, 1.5f),
        disVelX(-0.13f, 0.13f),disVelY(-0.1f, 0.1f)
{
    initialized_nvrtc = false;
    created_context = false;

    /// Checks for CUDA device availability. If no device is found,
    /// it sets a flag indicating CUDA is not available, forcing the use
    /// of CPU rendering (handled elsewhere, but noted here).
    /// If CUDA is available, it gets device properties to set the
    /// compute capability string for NVRTC compilation.
    /// Creates a default color palette (HSV).
    palette = createHSVPalette(BASIC_PALETTE_SIZE);
    paletteSize = BASIC_PALETTE_SIZE;
    /// Resizes the iteration points vector, used for drawing iteration paths.
    iterationpoints.resize(max_iterations);

    isCudaAvailable = true;
    int numDevices = 0;
    hipGetDeviceCount(&numDevices);
    if(numDevices == 0) {
        std::cout << "IMPORTANT NO AVAILABLE CUDA DEVICES FOUND" << std::endl;
        std::cout << "Forcing to use CPU rendering" << std::endl;
        std::cout << "Please make sure you have CUDA installed and your GPU supports it" << std::endl;
        isCudaAvailable = false;
        h_total_iterations = static_cast<unsigned int *>(malloc(sizeof(unsigned int)));
        compressed = static_cast<unsigned char *>(malloc(basic_width * basic_height * 4 * sizeof(unsigned char)));
        pixels = static_cast<unsigned char *>(malloc(basic_width * 2 * basic_height * 2 * 4 * sizeof(unsigned char)));
    }
    else {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, 0);
        compute_capability = "--gpu-architecture=compute_" + std::to_string(deviceProp.major) + std::to_string(deviceProp.minor);
        /// Allocates necessary memory on both host and device (GPU) for image data.
        ALLOCATE_ALL_IMAGE_MEMORY();
        /// Allocates necessary memory on both host and device (GPU) for non-image data (e.g., iteration counts).
        ALLOCATE_ALL_NON_IMAGE_MEMORY();
    }
}

/// Destructor for the FractalBase class.
/// Ensures proper cleanup of GPU resources and allocated memory.
template <typename Derived>
FractalBase<Derived>::~FractalBase() {
    if(!isCudaAvailable) {
        free(compressed);
        free(pixels);
        free(h_total_iterations);
    }
    else {
        /// Frees all allocated memory on both host and device (GPU).
        FREE_ALL_IMAGE_MEMORY();
        FREE_ALL_NON_IMAGE_MEMORY();
        /// Unloads the NVRTC module if it was loaded.
        if (module_loaded) CU_SAFE_CALL(hipModuleUnload(module));
        /// Destroys the CUDA context if it was created, using a macro
        /// to handle context switching if necessary.
        MAKE_CURR_CONTEXT_OPERATION(hipFree(nullptr), hipCtxDestroy(ctx), context);
    }
}

template <typename Derived>
unsigned int FractalBase<Derived>::get_max_iters() { return max_iterations; }

template <typename Derived>
bool FractalBase<Derived>::get_is_dragging() { return is_dragging; }

template <typename Derived>
void FractalBase<Derived>::set_max_iters(unsigned int max_iters) { max_iterations = max_iters; }

template <typename Derived>
double FractalBase<Derived>::get_x_offset() { return x_offset; }

template <typename Derived>
double FractalBase<Derived>::get_y_offset() { return y_offset; }

template <typename Derived>
double FractalBase<Derived>::get_zoom_x() { return zoom_x; }

template <typename Derived>
double FractalBase<Derived>::get_zoom_y() { return zoom_y; }

template <typename Derived>
double FractalBase<Derived>::get_zoom_scale() { return zoom_scale; }

template <typename Derived>
double FractalBase<Derived>::get_hardness_coeff() { return hardness_coeff; }

template <typename Derived>
sf::Texture FractalBase<Derived>::getTexture() { return texture; }

/// Sets the maximum computation capacity based on measured GPU performance (GFLOPS/GDFLOPS).
template <typename Derived>
void FractalBase<Derived>::setMaxComputation(float Gflops, float GDflops) { maxComputationF = 50.0f / 90 * Gflops; maxComputationD = 50.0f / 90 * GDflops; }

template <typename Derived>
Palletes FractalBase<Derived>::getPallete() { return curr_pallete; }

template <typename Derived>
sf::Vector2i FractalBase<Derived>::get_resolution() const { return {int(basic_width), int(basic_height)}; }

template <typename Derived>
unsigned int FractalBase<Derived>::get_compiling_percentage() {
    if (context == context_type::NVRTC) {
        return progress_compiling_percentage;
    }
    else {
        return 0;
    }
}

/// Sets the resolution of the fractal image.
/// When the resolution changes, the zoom and offset are recalculated
/// to keep the center of the previous view the same, maintaining
/// the visual focus point.
/// Existing GPU memory is freed and reallocated for the new resolution.
template <typename Derived>
void FractalBase<Derived>::set_resolution(sf::Vector2i target_resolution) {
    unsigned int old_width = width, old_height = height;

    width = basic_width = target_resolution.x;
    height = basic_height = target_resolution.y;

    double center_x = x_offset + (old_width / (zoom_x * zoom_scale)) / 2.0;
    double center_y = y_offset + (old_height / (zoom_y * zoom_scale)) / 2.0;

    zoom_x = basic_zoom_x * zoom_factor;
    zoom_y = basic_zoom_y * zoom_factor;

    x_offset = center_x - (width / (zoom_x * zoom_scale)) / 2.0;
    y_offset = center_y - (height / (zoom_y * zoom_scale)) / 2.0;

    FREE_ALL_IMAGE_MEMORY();
    ALLOCATE_ALL_IMAGE_MEMORY();
}


/// Resets the fractal view and parameters to their initial default state.
/// This includes resetting zoom, offset, iterations, and palette.
/// All existing GPU memory is freed and reallocated to ensure a clean state.
template <typename Derived>
void FractalBase<Derived>::reset() {
    // Free existing resources
    FREE_ALL_IMAGE_MEMORY();
    FREE_ALL_NON_IMAGE_MEMORY();

    INIT_BASIC_VALUES;

    if (std::is_same<Derived, fractals::julia>::value) {
        x_offset = 2.5;
        palette = createHSVPalette(BASIC_PALETTE_SIZE);
        paletteSize = BASIC_PALETTE_SIZE;
    }
    else {
        palette = createHSVPalette(BASIC_PALETTE_SIZE);
        paletteSize = BASIC_PALETTE_SIZE;
    }

    if(context == context_type::NVRTC) hipCtxSetCurrent(ctx);
    ALLOCATE_ALL_IMAGE_MEMORY();
    ALLOCATE_ALL_NON_IMAGE_MEMORY();

    iterationpoints.resize(max_iterations);
}

// that code served me good in the past, however it's being replaced with better version with atomic operations
// nevermind, atomic sucks!!! async is the way to go
// UwU sooooo saaaad UwU
//template <typename Derived>
//void FractalBase<Derived>::checkEventAndSetFlag(hipEvent_t event) {
//    while (hipEventQuery(event) == hipErrorNotReady) {
//        std::this_thread::sleep_for(std::chrono::milliseconds(1));
//    }
//    bool flag = false;
//    running_other_core = false;
//    hipMemcpy(stopFlagDevice, &flag, sizeof(bool), hipMemcpyHostToDevice);
//}

template <typename Derived>
context_type FractalBase<Derived>::get_context() { return context; }

template <typename Derived>
bool FractalBase<Derived>::get_bool_custom_formula() { return custom_formula; }

template class FractalBase<fractals::mandelbrot>;
template class FractalBase<fractals::julia>;