#include "hip/hip_runtime.h"
#include "FractalClass.cuh"
#include <hip/hip_runtime.h>
#include <TGUI/Backend/SFML-Graphics.hpp>
#include <iostream>
#include <thread>
#include <random>
#include <functional>
#include <fstream>

#define NVRTC_SAFE_CALL(x)                                        \
  do {                                                            \
    hiprtcResult result = x;                                       \
    if (result != HIPRTC_SUCCESS) {                                \
      std::cerr << "\nerror: " #x " failed with error "           \
                << hiprtcGetErrorString(result) << '\n';           \
      throw std::runtime_error("ERR");                                                     \
    }                                                             \
} while(0)

#define CU_SAFE_CALL(x)                                           \
  do {                                                            \
    hipError_t result = x;                                          \
    if (result != hipSuccess) {                                 \
      const char *msg;                                            \
      hipDrvGetErrorName(result, &msg);                               \
      std::cerr << "\nerror: " #x " failed with error "           \
                << msg << '\n';                                   \
      throw std::runtime_error(#x " failed with error " + std::string(msg));                                                     \
    }                                                             \
} while(0)

#define ALLOC_AND_COPY_TO_DEVICE_CU(cu_devPtr, hostVar, type, num)          \
  CU_SAFE_CALL(hipMalloc(&cu_devPtr, sizeof(type) * num));               \
  CU_SAFE_CALL(hipMemcpyHtoD(cu_devPtr, &hostVar, sizeof(type) * num));

#define CUDA_SAFE_CALL(x) \
  do {                    \
    hipError_t result = x;                                         \
    if (result != hipSuccess) {                                   \
      const char *msg =  hipGetErrorName(result);                  \
      std::cerr << "\nerror: " #x " failed with error "             \
                << msg << '\n';                                     \
      throw std::runtime_error(#x " failed with error " + std::string(msg));                                                       \
    }                                                               \
  }while(0)


#define MAKE_CURR_CONTEXT_OPERATION(x, y, ctx)                      \
  do {                                                              \
    if (context == context_type::CUDA){                             \
      CUDA_SAFE_CALL(x);                                            \
    }                                                               \
    else{                                                           \
      CU_SAFE_CALL(y);                                              \
    }                                                               \
} while(0)


#define COPY_PALETTE_TO_DEVICE(host, d, cu, ctx)                                                       \
  do {                                                                                          \
    if (context == context_type::CUDA){                                                         \
      CUDA_SAFE_CALL(hipMemcpy(d, host, sizeof(Color) * paletteSize, hipMemcpyHostToDevice));    \
    }                                                                                           \
    else{                                                                                       \
      CU_SAFE_CALL(hipMemcpyHtoD(cu, host, sizeof(Color) * paletteSize));                            \
    }                                                                                           \
} while(0)

#define ALLOCATE_ALL_IMAGE_MEMORY()                                                                                                                                                                 \
  do{                                                                                                                                                                                               \
    MAKE_CURR_CONTEXT_OPERATION(hipMalloc(&d_pixels, basic_width * 2 * basic_height * 2 * 4 * sizeof(unsigned char)), hipMalloc(&cu_d_pixels, sizeof(unsigned char) * basic_width * 2 * basic_height * 2 * 4), context);                  \
    MAKE_CURR_CONTEXT_OPERATION(hipHostMalloc(&pixels, basic_width * 2 * basic_height * 2 * 4 * sizeof(unsigned char)), hipHostAlloc((void**)&pixels, sizeof(unsigned char) * basic_width * 2 * basic_height * 2 * 4, 0), context);      \
    MAKE_CURR_CONTEXT_OPERATION(hipMalloc(&ssaa_buffer, basic_width * basic_height * 4 * sizeof(unsigned char)), hipMalloc(&CUssaa_buffer, basic_width * basic_height * 4 * sizeof(unsigned char)), context);                             \
    MAKE_CURR_CONTEXT_OPERATION(hipHostMalloc(&compressed, basic_width * basic_height * 4 * sizeof(unsigned char)), hipHostAlloc((void**)&compressed, basic_width * basic_height * 4 * sizeof(unsigned char), 0), context);                                                                      \
  } while(0)

#define FREE_ALL_IMAGE_MEMORY()                                                                 \
  do {                                                                                          \
    MAKE_CURR_CONTEXT_OPERATION(hipFree(d_pixels), hipFree(cu_d_pixels), context);           \
    MAKE_CURR_CONTEXT_OPERATION(hipFree(ssaa_buffer), hipFree(CUssaa_buffer), context);      \
    MAKE_CURR_CONTEXT_OPERATION(hipHostFree(pixels), hipHostFree(pixels), context);          \
    MAKE_CURR_CONTEXT_OPERATION(hipHostFree(compressed), hipHostFree(compressed), context);  \
  } while(0)

#define ALLOCATE_ALL_NON_IMAGE_MEMORY() \
  do {                                    \
    unsigned int zero = 0; \
    MAKE_CURR_CONTEXT_OPERATION(hipMalloc(&d_palette, palette.size() * sizeof(Color)), hipMalloc(&cu_palette, sizeof(Color) * paletteSize), context);\
    MAKE_CURR_CONTEXT_OPERATION(hipMemcpy(d_palette, palette.data(), palette.size() * sizeof(Color), hipMemcpyHostToDevice), hipMemcpyHtoD(cu_palette, palette.data(), sizeof(Color) * paletteSize), context);\
    MAKE_CURR_CONTEXT_OPERATION(hipStreamCreate(&stream), hipStreamCreateWithFlags(&CUss, 0), context);\
    MAKE_CURR_CONTEXT_OPERATION(hipMalloc(&d_total_iterations, sizeof(unsigned int)), hipMalloc(&cu_d_total_iterations, sizeof(unsigned int)), context);\
    MAKE_CURR_CONTEXT_OPERATION(hipMemset(d_total_iterations, 0, sizeof(unsigned int)), hipMemcpyHtoD(cu_d_total_iterations, &zero, sizeof(unsigned int)), context);\
    MAKE_CURR_CONTEXT_OPERATION(hipHostMalloc(&h_total_iterations , sizeof(unsigned int)), hipHostAlloc((void**)&h_total_iterations, sizeof(unsigned int), 0), context);\
    MAKE_CURR_CONTEXT_OPERATION(hipStreamCreate(&dataStream), hipStreamCreateWithFlags(&CUssData, 0), context);\
  } while(0)

#define FREE_ALL_NON_IMAGE_MEMORY() \
  do {                               \
    MAKE_CURR_CONTEXT_OPERATION(hipStreamDestroy(stream), hipStreamDestroy(CUss), context);\
    MAKE_CURR_CONTEXT_OPERATION(hipHostFree(h_total_iterations), hipHostFree(h_total_iterations), context);\
    MAKE_CURR_CONTEXT_OPERATION(hipStreamDestroy(dataStream), hipStreamDestroy(CUssData), context);\
    MAKE_CURR_CONTEXT_OPERATION(hipFree(d_total_iterations), hipFree(cu_d_total_iterations), context);\
    MAKE_CURR_CONTEXT_OPERATION(hipFree(d_palette), hipFree(cu_palette), context);\
  } while(0)


void cpu_render_mandelbrot(render_target target, unsigned char* pixels, unsigned int width, unsigned int height, double zoom_x, double zoom_y,
    double x_offset, double y_offset, Color* palette, unsigned int paletteSize,
    unsigned int max_iterations, unsigned int* total_iterations, std::atomic<unsigned char>& finish_flag
    )
{
    try {

        finish_flag.store(0);
        if (target.x_end > width) {
            target.x_end = width;
        }
        if (target.y_end > height) {
            target.y_end = height;
        }
        if (target.x_start < 0) {
            target.x_start = 0;
        }
        if (target.y_start < 0) {
            target.y_start = 0;
        }
        for(unsigned int y = target.y_start; y < target.y_end; ++y){
            for(unsigned int x = target.x_start; x < target.x_end; ++x){
                double zr = 0.0;
                double zi = 0.0;
                double cr = x / zoom_x - x_offset;
                double ci = y / zoom_y - y_offset;
                unsigned char r, g, b;
                float curr_iter = 0;
                while (curr_iter < max_iterations && zr * zr + zi * zi < 100.0) {
                    double tmp_zr = zr;
                    zr = zr * zr - zi * zi + cr;
                    zi = 2.0 * tmp_zr * zi + ci;

                    ++curr_iter;
                    if(finish_flag.load() == 2) {
                        finish_flag.store(1);
                        return;
                    }
                }
                if (curr_iter == max_iterations) {
                    r = g = b = 0;
                } else {
                    double smooth_iteration = curr_iter + 1.0f - log2f(log2f(sqrtf(zr * zr + zi * zi)));

                    const double cycle_scale_factor = 25.0f;
                    double virtual_pos = smooth_iteration * cycle_scale_factor;

                    double normalized_pingpong = fmodf(virtual_pos / static_cast<double>(paletteSize -1), 2.0f);
                    if (normalized_pingpong < 0.0f) {
                        normalized_pingpong += 2.0f;
                    }

                    double t_interp;
                    if (normalized_pingpong <= 1.0f) {
                        t_interp = normalized_pingpong;
                    } else {
                        t_interp = 2.0f - normalized_pingpong;
                    }

                    double float_index = t_interp * (paletteSize - 1);

                    int index1 = static_cast<int>(floorf(float_index));
                    int index2 = std::min(int(paletteSize - 1), index1 + 1);

                    index1 = std::max(0, index1);

                    double t_local = fmodf(float_index, 1.0f);
                    if (t_local < 0.0f) t_local += 1.0f;

                    Color color1 = getPaletteColor(index1, paletteSize, palette);
                    Color color2 = getPaletteColor(index2, paletteSize, palette);

                    float r_f = static_cast<float>(color1.r) + t_local * (static_cast<float>(color2.r) - static_cast<float>(color1.r));
                    float g_f = static_cast<float>(color1.g) + t_local * (static_cast<float>(color2.g) - static_cast<float>(color1.g));
                    float b_f = static_cast<float>(color1.b) + t_local * (static_cast<float>(color2.b) - static_cast<float>(color1.b));

                    r = static_cast<unsigned char>(std::max(0.0f, std::min(255.0f, r_f)));
                    g = static_cast<unsigned char>(std::max(0.0f, std::min(255.0f, g_f)));
                    b = static_cast<unsigned char>(std::max(0.0f, std::min(255.0f, b_f)));
                }
                const unsigned int index = (y * width + x) * 4;
                pixels[index] = r;
                pixels[index + 1] = g;
                pixels[index + 2] = b;
                pixels[index + 3] = 255;
                *total_iterations += curr_iter;
            }
        }
    }
    catch (const std::exception& e) {
        std::cerr << "ERROR in cpu_render_mandelbrot thread (target y=" << target.y_start << "): " << e.what() << std::endl;
    } catch (...) {
        std::cerr << "ERROR in cpu_render_mandelbrot thread (target y=" << target.y_start << "): Unknown exception!" << std::endl;
    }
    finish_flag.store(1);
}


template <typename Derived>
FractalBase<Derived>::FractalBase()
    :
    thread_stop_flags(std::thread::hardware_concurrency() * 100),
    max_iterations(300), basic_zoom_x(240.0), basic_zoom_y(240.0),
    zoom_x(basic_zoom_x), zoom_y(basic_zoom_y),
    x_offset(2.25), y_offset(1.25),
    zoom_factor(1.0), zoom_speed(0.1),
    zoom_scale(1.0),  maxComputationF(50.f), maxComputationD(50.f),
    basic_width(800), basic_height(600),
    width(800), height(600),
    sprite(texture), iterationline(sf::PrimitiveType::LineStrip),
    gen(rd()),
    disX(-2.f, 2.f), disY(-1.5f, 1.5f),
    disVelX(-0.13f, 0.13f),disVelY(-0.1f, 0.1f)
{
    initialized_nvrtc = false;
    created_context = false;

    isCudaAvailable = true;
    int numDevices;
    hipGetDeviceCount(&numDevices);
    if(numDevices == 0) {
        std::cout << "IMPORTANT NO AVAILABLE CUDA DEVICES FOUND" << std::endl;
        std::cout << "Forcing to use CPU rendering" << std::endl;
        std::cout << "Please make sure you have CUDA installed and your GPU supports it" << std::endl;
        isCudaAvailable = false;
    }
    palette = createHSVPalette(20000);
    paletteSize = 20000;


    ALLOCATE_ALL_IMAGE_MEMORY();
    ALLOCATE_ALL_NON_IMAGE_MEMORY();

    iterationpoints.resize(max_iterations);

}

template <typename Derived>
FractalBase<Derived>::~FractalBase() {
    FREE_ALL_IMAGE_MEMORY();
    FREE_ALL_NON_IMAGE_MEMORY();
    MAKE_CURR_CONTEXT_OPERATION(hipFree(nullptr), hipCtxDestroy(ctx), context);
}

template <typename Derived>
unsigned int FractalBase<Derived>::get_max_iters() { return max_iterations; }

template <typename Derived>
bool FractalBase<Derived>::get_is_dragging() { return is_dragging; }

template <typename Derived>
void FractalBase<Derived>::set_max_iters(unsigned int max_iters) { max_iterations = max_iters; }

template <typename Derived>
double FractalBase<Derived>::get_x_offset() { return x_offset; }

template <typename Derived>
double FractalBase<Derived>::get_y_offset() { return y_offset; }

template <typename Derived>
double FractalBase<Derived>::get_zoom_x() { return zoom_x; }

template <typename Derived>
double FractalBase<Derived>::get_zoom_y() { return zoom_y; }

template <typename Derived>
double FractalBase<Derived>::get_zoom_scale() { return zoom_scale; }

template <typename Derived>
double FractalBase<Derived>::get_hardness_coeff() { return hardness_coeff; }

template <typename Derived>
sf::Texture FractalBase<Derived>::getTexture() { return texture; }

template<typename Derived>
sf::Sprite FractalBase<Derived>::get_sprite_rect() { return sprite; }

template <typename Derived>
void FractalBase<Derived>::setMaxComputation(float Gflops, float GDflops) { maxComputationF = 50.0f / 90 * Gflops; maxComputationD = 50.0f / 90 * GDflops; }

template <typename Derived>
void FractalBase<Derived>::setPallete(std::string name) {
    if (name == "HSV") {
		palette = createHSVPalette(20000, degrees_offsetForHSV);
		paletteSize = 20000;
        COPY_PALETTE_TO_DEVICE(palette.data(), d_palette, cu_palette, context);
        curr_pallete = Palletes::HSV;
    }
    if (name == "Basic") {
		palette = BluePlusBlackWhitePalette(20000);
		paletteSize = 20000;
        COPY_PALETTE_TO_DEVICE(palette.data(), d_palette, cu_palette, context);
		curr_pallete = Palletes::Basic;
    }
    if (name == "BlackOWhite") {
		palette = CreateBlackOWhitePalette(20000);
		paletteSize = 20000;
        COPY_PALETTE_TO_DEVICE(palette.data(), d_palette, cu_palette, context);
		curr_pallete = Palletes::BlackOWhite;
    }
    if (name == "OscillatingGrayscale") {
		palette = CreateOscillatingGrayscalePalette(20000);
		paletteSize = 20000;
        COPY_PALETTE_TO_DEVICE(palette.data(), d_palette, cu_palette, context);
		curr_pallete = Palletes::OscillatingGrayscale;
    }
    if (name == "Interpolated") {
		palette = CreateInterpolatedPalette(20000);
		paletteSize = 20000;
        COPY_PALETTE_TO_DEVICE(palette.data(), d_palette, cu_palette, context);
		curr_pallete = Palletes::Interpolated;
    }
    if (name == "Pastel") {
        palette = CreatePastelPalette(20000);
        paletteSize = 20000;
        COPY_PALETTE_TO_DEVICE(palette.data(), d_palette, cu_palette, context);
        curr_pallete = Palletes::Pastel;
    }
    if (name == "CyclicHSV") {
		palette = CreateCyclicHSVPpalette(20000);
		paletteSize = 20000;
        COPY_PALETTE_TO_DEVICE(palette.data(), d_palette, cu_palette, context);
		curr_pallete = Palletes::CyclicHSV;
    }
    if (name == "Fire") {
		palette = CreateFirePalette(20000);
		paletteSize = 20000;
        COPY_PALETTE_TO_DEVICE(palette.data(), d_palette, cu_palette, context);
		curr_pallete = Palletes::Fire;
    }
    if (name == "FractalPattern") {
		palette = CreateFractalPatternPalette(20000);
		paletteSize = 20000;
        COPY_PALETTE_TO_DEVICE(palette.data(), d_palette, cu_palette, context);
		curr_pallete = Palletes::FractalPattern;
    }
    if (name == "PerlinNoise") {
		palette = CreatePerlinNoisePalette(20000);
		paletteSize = 20000;
        COPY_PALETTE_TO_DEVICE(palette.data(), d_palette, cu_palette, context);
		curr_pallete = Palletes::PerlinNoise;
    }
    if (name == "Water") {
		palette = CreateWaterPalette(20000);
		paletteSize = 20000;
        COPY_PALETTE_TO_DEVICE(palette.data(), d_palette, cu_palette, context);
		curr_pallete = Palletes::Water;
    }
    if (name == "Sunset") {
		palette = CreateSunsetPalette(20000);
		paletteSize = 20000;
        COPY_PALETTE_TO_DEVICE(palette.data(), d_palette, cu_palette, context);
		curr_pallete = Palletes::Sunset;
    }
    if (name == "DeepSpace") {
		palette = CreateDeepSpaceWideVeinsPalette(20000);
		paletteSize = 20000;
        COPY_PALETTE_TO_DEVICE(palette.data(), d_palette, cu_palette, context);
		curr_pallete = Palletes::DeepSpace;
    }
    if (name == "Physchodelic") {
		palette = CreatePsychedelicWavePalette(20000);
		paletteSize = 20000;
        COPY_PALETTE_TO_DEVICE(palette.data(), d_palette, cu_palette, context);
		curr_pallete = Palletes::Physchodelic;
    }
    if (name == "IceCave") {
		palette = CreateIceCavePalette(20000);
		paletteSize = 20000;
        COPY_PALETTE_TO_DEVICE(palette.data(), d_palette, cu_palette, context);
		curr_pallete = Palletes::IceCave;
    }
    if (name == "AccretionDisk") {
		palette = CreateAccretionDiskPalette(20000);
		paletteSize = 20000;
        COPY_PALETTE_TO_DEVICE(palette.data(), d_palette, cu_palette, context);
		curr_pallete = Palletes::AccretionDisk;
    }
    if (name == "ElectricNebula") {
		palette = CreateElectricNebulaPalette(20000);
		paletteSize = 20000;
        COPY_PALETTE_TO_DEVICE(palette.data(), d_palette, cu_palette, context);
		curr_pallete = Palletes::ElectricNebula;
    }
    if (name == "Random") {
        palette = CreateRandomPalette(20000);
        paletteSize = 20000;
        COPY_PALETTE_TO_DEVICE(palette.data(), d_palette, cu_palette, context);
        curr_pallete = Palletes::Random;
    }
}

template <typename Derived>
Palletes FractalBase<Derived>::getPallete() { return curr_pallete; }

template <typename Derived>
sf::Vector2i FractalBase<Derived>::get_resolution() const { return {int(basic_width), int(basic_height)}; }

template <typename Derived>
void FractalBase<Derived>::SetDegreesOffsetForHSV(int degrees) { degrees_offsetForHSV = degrees; setPallete("HSV"); }

template <typename Derived>
void FractalBase<Derived>::set_resolution(sf::Vector2i target_resolution) {
    unsigned int old_width = width, old_height = height;

    width = basic_width = target_resolution.x;
    height = basic_height = target_resolution.y;

    double center_x = x_offset + (old_width / (zoom_x * zoom_scale)) / 2.0;
    double center_y = y_offset + (old_height / (zoom_y * zoom_scale)) / 2.0;

    zoom_x = basic_zoom_x * zoom_factor;
    zoom_y = basic_zoom_y * zoom_factor;

    x_offset = center_x - (width / (zoom_x * zoom_scale)) / 2.0;
    y_offset = center_y - (height / (zoom_y * zoom_scale)) / 2.0;

    FREE_ALL_IMAGE_MEMORY();
    ALLOCATE_ALL_IMAGE_MEMORY();
}


template <typename Derived>
void FractalBase<Derived>::post_processing() {
    if(!isCudaAvailable){
        image.resize({ basic_width, basic_height }, pixels);
    }
    else if (antialiasing) {
        // SSAA rendering
        dim3 dimBlock(32, 32);
        dim3 dimGrid(
            (width + dimBlock.x - 1) / dimBlock.x,
            (height + dimBlock.y - 1) / dimBlock.y
        );
        if(!custom_formula){
            ANTIALIASING_SSAA4<<<dimGrid, dimBlock, 0, stream>>>(d_pixels, ssaa_buffer, basic_width * 2, basic_height * 2, basic_width, basic_height);
            hipMemcpyAsync(compressed, ssaa_buffer, basic_width * basic_height * 4 * sizeof(unsigned char), hipMemcpyDeviceToHost, stream);
            hipStreamSynchronize(stream);
        }
        else {
            CU_SAFE_CALL(hipCtxSetCurrent(ctx));
            void* params[] = { &cu_d_pixels, &CUssaa_buffer, &width, &height, &basic_width, &basic_height };
            CU_SAFE_CALL(hipModuleLaunchKernel(kernelAntialiasing,
                    dimGrid.x, dimGrid.y, 1,
                    dimBlock.x, dimBlock.y, 1,
                    0,
                    CUss,
                    params,
                    nullptr
                    ));
            CU_SAFE_CALL(hipMemcpyDtoHAsync((void**)compressed, CUssaa_buffer, basic_height * basic_width * 4 * sizeof(unsigned char),CUss));
            CU_SAFE_CALL(hipStreamSynchronize(CUss));
        }

        image.resize({ basic_width, basic_height }, compressed);

    }
    else {
        if(custom_formula) {
            CU_SAFE_CALL(hipCtxSetCurrent(ctx));
            CU_SAFE_CALL(hipMemcpyDtoHAsync(pixels, cu_d_pixels, basic_width * basic_height * 4 * sizeof(unsigned char), CUss));
            CU_SAFE_CALL(hipMemcpyDtoHAsync(h_total_iterations, cu_d_total_iterations, sizeof(int), CUssData));
            CU_SAFE_CALL(hipStreamSynchronize(CUss));
        }
        else {
            hipMemcpyAsync(pixels, d_pixels, width * height * 4 * sizeof(unsigned char), hipMemcpyDeviceToHost, stream);
            hipMemcpyAsync(h_total_iterations, d_total_iterations, sizeof(int), hipMemcpyDeviceToHost, dataStream);
            // THERE AIN'T NO WAY I AM DOING THAT SYNC LOGIC, JUST LET IT BE
            hipStreamSynchronize(stream);
        }
        hardness_coeff = *h_total_iterations / (width * height * 1.0);
        image.resize({ basic_width, basic_height }, pixels);
    }
    if(!texture.loadFromImage(image, true)) {
        std::cerr << "Data corrupted!\n";
    }
    sprite.setTexture(texture, true);
    sprite.setPosition({ 0, 0 });
}

// To surely not forget anything lets make sure to delete everything and reallocate
template <typename Derived>
void FractalBase<Derived>::reset() {
    // Free existing resources
    FREE_ALL_IMAGE_MEMORY();
    FREE_ALL_NON_IMAGE_MEMORY();

    // Reset parameters
    max_iterations = 300;
    basic_zoom_x = 240.0;
    basic_zoom_y = 240.0;
    zoom_x = basic_zoom_x;
    zoom_y = basic_zoom_y;
    x_offset = 3.0;
    y_offset = 1.85;
    zoom_factor = 1.0;
    zoom_speed = 0.1;
    zoom_scale = 1.0;
    width = 800;
    height = 600;
    basic_width = 800;
    basic_height = 600;

    if (std::is_same<Derived, fractals::julia>::value) {
        x_offset = 2.5;
        palette = createHSVPalette(20000);
        paletteSize = 20000;
    }
    else {
        palette = createHSVPalette(20000);
        paletteSize = 20000;
    }

    if(context == context_type::NVRTC) hipCtxSetCurrent(ctx);
    ALLOCATE_ALL_IMAGE_MEMORY();
    ALLOCATE_ALL_NON_IMAGE_MEMORY();

    iterationpoints.resize(max_iterations);
}

// that code served me good in the past, however it's being replaced with better version with atomic operations
// nevermind, atomic sucks!!! async is the way to go
// UwU sooooo saaaad UwU
//template <typename Derived>
//void FractalBase<Derived>::checkEventAndSetFlag(hipEvent_t event) {
//    while (hipEventQuery(event) == hipErrorNotReady) {
//        std::this_thread::sleep_for(std::chrono::milliseconds(1));
//    }
//    bool flag = false;
//    running_other_core = false;
//    hipMemcpy(stopFlagDevice, &flag, sizeof(bool), hipMemcpyHostToDevice);
//}

/// Formula should be like this one\n
/// new_real = z_real * z_real - z_imag * z_imag + real;\n
/// z_imag =  2 * z_real * z_imag + imag;
/// BEFORE USING THIS FUNCTION MAKE SURE TO SET THE CONTEXT TO NVRTC
template <>
void FractalBase<fractals::mandelbrot>::set_custom_formula(const std::string formula) {
    if(!custom_formula) {
        set_context(context_type::NVRTC);
    }
    CU_SAFE_CALL(hipCtxSetCurrent(ctx));
    kernel_code = R"(
#include "../include/fractals/custom.cuh"
template <typename T>
__global__ void fractal_rendering(
        unsigned char* pixels, unsigned long size_of_pixels, unsigned int width, unsigned int height,
        T zoom_x, T zoom_y, T x_offset, T y_offset,
        Color* d_palette, unsigned int paletteSize, T maxIterations, unsigned int* d_total_iterations)
{
    const unsigned int x =   blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int y =   blockIdx.y * blockDim.y + threadIdx.y;
    const unsigned int id = threadIdx.y * blockDim.x + threadIdx.x;
    if (x == 0 && y == 0) {
        *d_total_iterations = 0;
    }

    const size_t expected_size = width * height * 4;

    const T scale_factor = static_cast<T>(size_of_pixels) / static_cast<T>(expected_size);

    if (x < width && y < height) {
        __shared__ unsigned int total_iterations[1024];
        const T real = x / zoom_x - x_offset;
        const T imag = y / zoom_y - y_offset;
        T new_real = 0.0;
        T z_real = 0.0;
        T z_imag = 0.0;
        T current_iteration = 0;
        T z_comp = z_real * z_real + z_imag * z_imag;

        while (z_comp < 4 && current_iteration < maxIterations) {
)" + formula + R"(
            z_real = new_real;
            z_comp = z_real * z_real + z_imag * z_imag;
            current_iteration++;
        }

        total_iterations[id] = static_cast<unsigned int>(current_iteration);
//        __syncthreads();

        for (unsigned int s = blockDim.x * blockDim.y / 2; s > 0; s >>= 1) {
            if (id < s) {
                total_iterations[id] += total_iterations[id + s];
            }
//            __syncthreads();
        }
        if (id == 0) {
            //d_total_iterations += total_iterations[0];
            atomicAdd(d_total_iterations, total_iterations[0]);
        }

        unsigned char r, g, b;
        if (current_iteration == maxIterations) {
            r = g = b = 0;
        }

        else {

            //modulus = hypot(z_real, z_imag);
            //double escape_radius = 2.0;
            //if (modulus > escape_radius) {
            //    double nu = log2(log2(modulus) - log2(escape_radius));
            //    current_iteration = current_iteration + 1 - nu;
            //}
            T smooth_iteration = current_iteration + 1.0f - log2f(log2f(sqrtf(z_real * z_real + z_imag * z_imag)));

            const T cycle_scale_factor = 25.0f;
            T virtual_pos = smooth_iteration * cycle_scale_factor;

            T normalized_pingpong = fmodf(virtual_pos / static_cast<T>(paletteSize -1), 2.0f);
            if (normalized_pingpong < 0.0f) {
                normalized_pingpong += 2.0f;
            }

            T t_interp;
            if (normalized_pingpong <= 1.0f) {
                t_interp = normalized_pingpong;
            } else {
                t_interp = 2.0f - normalized_pingpong;
            }

            T float_index = t_interp * (paletteSize - 1);

            int index1 = static_cast<int>(floorf(float_index));
            int index2 = min(paletteSize - 1, index1 + 1);

            index1 = max(0, index1);

            T t_local = fmodf(float_index, 1.0f);
            if (t_local < 0.0f) t_local += 1.0f;

            Color color1 = getPaletteColor(index1, paletteSize, d_palette);
            Color color2 = getPaletteColor(index2, paletteSize, d_palette);

            float r_f = static_cast<float>(color1.r) + t_local * (static_cast<float>(color2.r) - static_cast<float>(color1.r));
            float g_f = static_cast<float>(color1.g) + t_local * (static_cast<float>(color2.g) - static_cast<float>(color1.g));
            float b_f = static_cast<float>(color1.b) + t_local * (static_cast<float>(color2.b) - static_cast<float>(color1.b));

            r = static_cast<unsigned char>(max(0.0f, min(255.0f, r_f)));
            g = static_cast<unsigned char>(max(0.0f, min(255.0f, g_f)));
            b = static_cast<unsigned char>(max(0.0f, min(255.0f, b_f)));
        }
        const unsigned int base_index = (y * width + x) * 4;
        for (int i = 0; i < scale_factor * 4; i += 4) {
            const unsigned int index = base_index + i;
            pixels[index] = r;
            pixels[index + 1] = g;
            pixels[index + 2] = b;
            pixels[index + 3] = 255;
        }
    }
}

template __global__ void fractal_rendering<float>(
        unsigned char*, size_t, unsigned int, unsigned int,
        float, float, float, float,
        Color*, unsigned int, float, unsigned int*);

template __global__ void fractal_rendering<double>(
        unsigned char*, size_t, unsigned int, unsigned int,
        double, double, double, double,
        Color*, unsigned int, double, unsigned int*);

__global__
void ANTIALIASING_SSAA4(unsigned char* src, unsigned char* dest, unsigned int src_width, unsigned int src_height, unsigned int dest_width, unsigned int dest_height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < dest_width && y < dest_height) {
        int r = 0, g = 0, b = 0;
        for (int i = 0; i < 2; ++i) {
            for (int j = 0; j < 2; ++j) {
                int src_x = x * 2 + i;
                int src_y = y * 2 + j;
                if (src_x < src_width && src_y < src_height) {
                    int src_index = (src_y * src_width + src_x) * 4;
                    r += src[src_index];
                    g += src[src_index + 1];
                    b += src[src_index + 2];
                }
            }
        }
        int dest_index = (y * dest_width + x) * 4;
        dest[dest_index] = r / 4;
        dest[dest_index + 1] = g / 4;
        dest[dest_index + 2] = b / 4;
        dest[dest_index + 3] = 255;
    }
}
)";
    hiprtcProgram prog = nullptr;
    hiprtcResult compileResult;
    std::string lowered_kernel_name_float_str;
    std::string lowered_kernel_name_double_str;
    std::string lowered_kernel_name_ssaa_str;

    try {
        const std::ifstream header_file("../include/fractals/custom.cuh");
        if (!header_file.is_open()) {
            throw std::runtime_error("Could not open header file: custom.cuh");
        }

        std::stringstream buffer;
        buffer << header_file.rdbuf();
        const std::string header_content = buffer.str();

        const char *header_data = header_content.c_str();
        constexpr char *header_name = "custom.cuh";
        const char *headers[] = {header_data};
        const char *includeNames[] = {header_name};

        NVRTC_SAFE_CALL(hiprtcCreateProgram(&prog, kernel_code.c_str(), "custom.cu", 1, headers, includeNames));

        NVRTC_SAFE_CALL(hiprtcAddNameExpression(prog, "fractal_rendering<float>"));
        NVRTC_SAFE_CALL(hiprtcAddNameExpression(prog, "fractal_rendering<double>"));
        NVRTC_SAFE_CALL(hiprtcAddNameExpression(prog, "ANTIALIASING_SSAA4"));

        std::vector<const char*> compile_options;
        const char* vcpkg_root = std::getenv("VCPKG_ROOT");
        if (vcpkg_root == nullptr) {
            std::cerr << "Warning: VCPKG_ROOT environment variable not set. Assuming a default." << std::endl;
            vcpkg_root = "/home/progamers/vcpkg";
        }
        const std::string triplet = "x64-linux";
        std::string sfml_include_path = "-I" + std::string(vcpkg_root) + "/installed/" + triplet + "/include";
        compile_options.push_back(sfml_include_path.c_str());

        std::string system_include_path1 = "-I/usr/include/c++/14.2.1";
        compile_options.push_back(system_include_path1.c_str());

        std::string system_include_path2 = "-I/usr/include/c++/14.2.1/x86_64-pc-linux-gnu";
        compile_options.push_back(system_include_path2.c_str());

        compile_options.push_back("--gpu-architecture=compute_75");

        const char** opts = compile_options.data();
        int num_opts = compile_options.size();

        compileResult = hiprtcCompileProgram(prog, num_opts, opts);

        size_t logSize = 0;
        if (prog) {
            NVRTC_SAFE_CALL(hiprtcGetProgramLogSize(prog, &logSize));
        }

        std::string log;
        if (logSize > 1) {
            log.resize(logSize);
            NVRTC_SAFE_CALL(hiprtcGetProgramLog(prog, &log[0]));
        }

        if (compileResult != HIPRTC_SUCCESS) {
            std::cerr << "---------------------\n";
            std::cerr << "NVRTC Compilation Failed:\n";
            std::cerr << "Result Code: " << hiprtcGetErrorString(compileResult) << "\n";
            std::cerr << "---------------------\n";
            std::cerr << "Compilation Log:\n";
            std::cerr << log << std::endl;
            std::cerr << "---------------------\n";

            if (prog) {
                hiprtcDestroyProgram(&prog);
            }
            return;
        } else {
            std::cout << "NVRTC Compilation Succeeded.\n";
            if (!log.empty() && log.length() > 1) {
                std::cout << "---------------------\n";
                std::cout << "Compilation Log (Warnings/Info):\n";
                std::cout << log << std::endl;
                std::cout << "---------------------\n";
            }
        }

        const char* lowered_name_float_ptr = nullptr;
        if (prog) {
            NVRTC_SAFE_CALL(hiprtcGetLoweredName(prog, "fractal_rendering<float>", &lowered_name_float_ptr));
        }

        if (lowered_name_float_ptr) {
            lowered_kernel_name_float_str = lowered_name_float_ptr;
        } else {
            // Handle error if name not found after successful compilation (e.g., return)
            std::cerr << "Error: Could not get lowered name for fractal_rendering<float>\n";
            if (prog) hiprtcDestroyProgram(&prog); return;
        }

        const char* lowered_name_double_ptr = nullptr;
        if (prog) {
            NVRTC_SAFE_CALL(hiprtcGetLoweredName(prog, "fractal_rendering<double>", &lowered_name_double_ptr));
        }

        if (lowered_name_double_ptr) {
            lowered_kernel_name_double_str = lowered_name_double_ptr;
        } else {
            std::cerr << "Error: Could not get lowered name for fractal_rendering<double>\n";
            if (prog) hiprtcDestroyProgram(&prog); return;
        }

        const char* lowered_name_ssaa_ptr = nullptr;
        if (prog) {
            NVRTC_SAFE_CALL(hiprtcGetLoweredName(prog, "ANTIALIASING_SSAA4", &lowered_name_ssaa_ptr));
        }

        if (lowered_name_ssaa_ptr) {
            lowered_kernel_name_ssaa_str = lowered_name_ssaa_ptr;
        } else {
            std::cerr << "Error: Could not get lowered name for ANTIALIASING_SSAA4\n";
            if (prog) hiprtcDestroyProgram(&prog); return;
        }


        size_t ptxSize = 0;
        if (prog) {
            NVRTC_SAFE_CALL(hiprtcGetCodeSize(prog, &ptxSize));
        }

        std::vector<char> ptx;
        if (ptxSize > 0 && prog) {
            ptx.resize(ptxSize);
            NVRTC_SAFE_CALL(hiprtcGetCode(prog, ptx.data()));
        }

        if (prog) {
            NVRTC_SAFE_CALL(hiprtcDestroyProgram(&prog));
            prog = nullptr;
        }

        if (module_loaded) {
            CU_SAFE_CALL(hipModuleUnload(module));
            module = nullptr;
            module_loaded = false;
        }

        if (!ptx.empty()) {
            CU_SAFE_CALL(hipModuleLoadDataEx(&module, ptx.data(), 0, 0, 0));
            module_loaded = true;
        }

        if (module && !lowered_kernel_name_float_str.empty()) {
            CU_SAFE_CALL(hipModuleGetFunction(&kernelFloat, module, lowered_kernel_name_float_str.c_str()));
        } else if (!module) { /* Module not loaded */ } else { /* Name string empty, handle as error */ }


        if (module && !lowered_kernel_name_double_str.empty()) {
            CU_SAFE_CALL(hipModuleGetFunction(&kernelDouble, module, lowered_kernel_name_double_str.c_str()));
        } else if (!module) { /* Module not loaded */ } else { /* Name string empty, handle as error */ }


        if (module && !lowered_kernel_name_ssaa_str.empty()) {
            CU_SAFE_CALL(hipModuleGetFunction(&kernelAntialiasing, module, lowered_kernel_name_ssaa_str.c_str()));
        } else if (!module) { /* Module not loaded */ } else { /* Name string empty, handle as error */ }


    } catch (const std::exception& e) {
        std::cerr << "Exception in NVRTC setup/compilation/loading: " << e.what() << std::endl;
        if (prog) {
            NVRTC_SAFE_CALL(hiprtcDestroyProgram(&prog));
        }
        throw;
    }
    std::cout << "Kernel loaded successfully.\n";
    custom_formula = true;
}


template <>
/// Formula should be like this one\n
/// new_real = z_real * z_real - z_imag * z_imag + real;\n
/// z_imag =  2 * z_real * z_imag + imag;
/// BEFORE USING THIS FUNCTION MAKE SURE TO SET THE CONTEXT TO NVRTC
void FractalBase<fractals::julia>::set_custom_formula(const std::string formula) {
    if(!custom_formula) {
        set_context(context_type::NVRTC);
    }
    CU_SAFE_CALL(hipCtxSetCurrent(ctx));
    kernel_code = R"(
#include "../include/fractals/custom.cuh"
template <typename T>
__global__ void fractal_rendering_julia(
        unsigned char* pixels, unsigned long size_of_pixels, unsigned int width, unsigned int height,
        T zoom_x, T zoom_y, T x_offset, T y_offset,
        Color* d_palette, unsigned int paletteSize, T maxIterations, unsigned int* d_total_iterations, T cReal, T cImaginary)
{
    const unsigned int x =   blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int y =   blockIdx.y * blockDim.y + threadIdx.y;
    const unsigned int id = threadIdx.y * blockDim.x + threadIdx.x;
    if (x == 0 && y == 0) {
        *d_total_iterations = 0;
    }

    const size_t expected_size = width * height * 4;

    const T scale_factor = static_cast<T>(size_of_pixels) / static_cast<T>(expected_size);

    if (x < width && y < height) {
        __shared__ unsigned int total_iterations[1024];
        T z_real = x / zoom_x - x_offset;
        T z_imag = y / zoom_y - y_offset;
        T real = cReal;
        T imag = cImaginary;
        T new_real = 0.0;
        T current_iteration = 0;
        T z_comp = z_real * z_real + z_imag * z_imag;

        while (z_comp < 4 && current_iteration < maxIterations) {
)" + formula + R"(
            z_real = new_real;
            z_comp = z_real * z_real + z_imag * z_imag;
            current_iteration++;
        }

        total_iterations[id] = static_cast<unsigned int>(current_iteration);
//        __syncthreads();

        for (unsigned int s = blockDim.x * blockDim.y / 2; s > 0; s >>= 1) {
            if (id < s) {
                total_iterations[id] += total_iterations[id + s];
            }
//            __syncthreads();
        }
        if (id == 0) {
            //d_total_iterations += total_iterations[0];
            atomicAdd(d_total_iterations, total_iterations[0]);
        }

        unsigned char r, g, b;
        if (current_iteration == maxIterations) {
            r = g = b = 0;
        }

        else {

            //modulus = hypot(z_real, z_imag);
            //double escape_radius = 2.0;
            //if (modulus > escape_radius) {
            //    double nu = log2(log2(modulus) - log2(escape_radius));
            //    current_iteration = current_iteration + 1 - nu;
            //}
            T smooth_iteration = current_iteration + 1.0f - log2f(log2f(sqrtf(z_real * z_real + z_imag * z_imag)));

            const T cycle_scale_factor = 25.0f;
            T virtual_pos = smooth_iteration * cycle_scale_factor;

            T normalized_pingpong = fmodf(virtual_pos / static_cast<T>(paletteSize -1), 2.0f);
            if (normalized_pingpong < 0.0f) {
                normalized_pingpong += 2.0f;
            }

            T t_interp;
            if (normalized_pingpong <= 1.0f) {
                t_interp = normalized_pingpong;
            } else {
                t_interp = 2.0f - normalized_pingpong;
            }

            T float_index = t_interp * (paletteSize - 1);

            int index1 = static_cast<int>(floorf(float_index));
            int index2 = min(paletteSize - 1, index1 + 1);

            index1 = max(0, index1);

            T t_local = fmodf(float_index, 1.0f);
            if (t_local < 0.0f) t_local += 1.0f;

            Color color1 = getPaletteColor(index1, paletteSize, d_palette);
            Color color2 = getPaletteColor(index2, paletteSize, d_palette);

            float r_f = static_cast<float>(color1.r) + t_local * (static_cast<float>(color2.r) - static_cast<float>(color1.r));
            float g_f = static_cast<float>(color1.g) + t_local * (static_cast<float>(color2.g) - static_cast<float>(color1.g));
            float b_f = static_cast<float>(color1.b) + t_local * (static_cast<float>(color2.b) - static_cast<float>(color1.b));

            r = static_cast<unsigned char>(max(0.0f, min(255.0f, r_f)));
            g = static_cast<unsigned char>(max(0.0f, min(255.0f, g_f)));
            b = static_cast<unsigned char>(max(0.0f, min(255.0f, b_f)));
        }
        const unsigned int base_index = (y * width + x) * 4;
        for (int i = 0; i < scale_factor * 4; i += 4) {
            const unsigned int index = base_index + i;
            pixels[index] = r;
            pixels[index + 1] = g;
            pixels[index + 2] = b;
            pixels[index + 3] = 255;
        }
    }
}

template __global__ void fractal_rendering_julia<float>(
        unsigned char*, size_t, unsigned int, unsigned int,
        float, float, float, float,
        Color*, unsigned int, float, unsigned int*, float, float);

template __global__ void fractal_rendering_julia<double>(
        unsigned char*, size_t, unsigned int, unsigned int,
        double, double, double, double,
        Color*, unsigned int, double, unsigned int*, double, double);

__global__
void ANTIALIASING_SSAA4(unsigned char* src, unsigned char* dest, unsigned int src_width, unsigned int src_height, unsigned int dest_width, unsigned int dest_height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < dest_width && y < dest_height) {
        int r = 0, g = 0, b = 0;
        for (int i = 0; i < 2; ++i) {
            for (int j = 0; j < 2; ++j) {
                int src_x = x * 2 + i;
                int src_y = y * 2 + j;
                if (src_x < src_width && src_y < src_height) {
                    int src_index = (src_y * src_width + src_x) * 4;
                    r += src[src_index];
                    g += src[src_index + 1];
                    b += src[src_index + 2];
                }
            }
        }
        int dest_index = (y * dest_width + x) * 4;
        dest[dest_index] = r / 4;
        dest[dest_index + 1] = g / 4;
        dest[dest_index + 2] = b / 4;
        dest[dest_index + 3] = 255;
    }
}
)";
    hiprtcProgram prog = nullptr;
    hiprtcResult compileResult;
    std::string lowered_kernel_name_float_str;
    std::string lowered_kernel_name_double_str;
    std::string lowered_kernel_name_ssaa_str;

    try {
        const std::ifstream header_file("../include/fractals/custom.cuh");
        if (!header_file.is_open()) {
            throw std::runtime_error("Could not open header file: custom.cuh");
        }

        std::stringstream buffer;
        buffer << header_file.rdbuf();
        const std::string header_content = buffer.str();

        const char *header_data = header_content.c_str();
        constexpr char *header_name = "custom.cuh";
        const char *headers[] = {header_data};
        const char *includeNames[] = {header_name};

        NVRTC_SAFE_CALL(hiprtcCreateProgram(&prog, kernel_code.c_str(), "custom.cu", 1, headers, includeNames));

        NVRTC_SAFE_CALL(hiprtcAddNameExpression(prog, "fractal_rendering_julia<float>"));
        NVRTC_SAFE_CALL(hiprtcAddNameExpression(prog, "fractal_rendering_julia<double>"));
        NVRTC_SAFE_CALL(hiprtcAddNameExpression(prog, "ANTIALIASING_SSAA4"));

        std::vector<const char*> compile_options;
        const char* vcpkg_root = std::getenv("VCPKG_ROOT");
        if (vcpkg_root == nullptr) {
            std::cerr << "Warning: VCPKG_ROOT environment variable not set. Assuming a default." << std::endl;
            vcpkg_root = "/home/progamers/vcpkg";
        }
        const std::string triplet = "x64-linux";
        std::string sfml_include_path = "-I" + std::string(vcpkg_root) + "/installed/" + triplet + "/include";
        compile_options.push_back(sfml_include_path.c_str());

        std::string system_include_path1 = "-I/usr/include/c++/14.2.1";
        compile_options.push_back(system_include_path1.c_str());

        std::string system_include_path2 = "-I/usr/include/c++/14.2.1/x86_64-pc-linux-gnu";
        compile_options.push_back(system_include_path2.c_str());

        compile_options.push_back("--gpu-architecture=compute_75");

        const char** opts = compile_options.data();
        int num_opts = compile_options.size();

        compileResult = hiprtcCompileProgram(prog, num_opts, opts);

        size_t logSize = 0;
        if (prog) {
            NVRTC_SAFE_CALL(hiprtcGetProgramLogSize(prog, &logSize));
        }

        std::string log;
        if (logSize > 1) {
            log.resize(logSize);
            NVRTC_SAFE_CALL(hiprtcGetProgramLog(prog, &log[0]));
        }

        if (compileResult != HIPRTC_SUCCESS) {
            std::cerr << "---------------------\n";
            std::cerr << "NVRTC Compilation Failed:\n";
            std::cerr << "Result Code: " << hiprtcGetErrorString(compileResult) << "\n";
            std::cerr << "---------------------\n";
            std::cerr << "Compilation Log:\n";
            std::cerr << log << std::endl;
            std::cerr << "---------------------\n";

            if (prog) {
                NVRTC_SAFE_CALL(hiprtcDestroyProgram(&prog));
            }
            return;
        } else {
            std::cout << "NVRTC Compilation Succeeded.\n";
            if (!log.empty() && log.length() > 1) {
                std::cout << "---------------------\n";
                std::cout << "Compilation Log (Warnings/Info):\n";
                std::cout << log << std::endl;
                std::cout << "---------------------\n";
            }
        }

        const char* lowered_name_float_ptr = nullptr;
        if (prog) {
            NVRTC_SAFE_CALL(hiprtcGetLoweredName(prog, "fractal_rendering_julia<float>", &lowered_name_float_ptr));
        }

        if (lowered_name_float_ptr) {
            lowered_kernel_name_float_str = lowered_name_float_ptr;
        } else {
            // Handle error if name not found after successful compilation (e.g., return)
            std::cerr << "Error: Could not get lowered name for fractal_rendering_julia<float>\n";
            if (prog) NVRTC_SAFE_CALL(hiprtcDestroyProgram(&prog)); return;
        }

        const char* lowered_name_double_ptr = nullptr;
        if (prog) {
            NVRTC_SAFE_CALL(hiprtcGetLoweredName(prog, "fractal_rendering_julia<double>", &lowered_name_double_ptr));
        }

        if (lowered_name_double_ptr) {
            lowered_kernel_name_double_str = lowered_name_double_ptr;
        } else {
            std::cerr << "Error: Could not get lowered name for fractal_rendering_julia<double>\n";
            if (prog) NVRTC_SAFE_CALL(hiprtcDestroyProgram(&prog)); return;
        }

        const char* lowered_name_ssaa_ptr = nullptr;
        if (prog) {
            NVRTC_SAFE_CALL(hiprtcGetLoweredName(prog, "ANTIALIASING_SSAA4", &lowered_name_ssaa_ptr));
        }

        if (lowered_name_ssaa_ptr) {
            lowered_kernel_name_ssaa_str = lowered_name_ssaa_ptr;
        } else {
            std::cerr << "Error: Could not get lowered name for ANTIALIASING_SSAA4\n";
            if (prog) NVRTC_SAFE_CALL(hiprtcDestroyProgram(&prog)); return;
        }


        size_t ptxSize = 0;
        if (prog) {
            NVRTC_SAFE_CALL(hiprtcGetCodeSize(prog, &ptxSize));
        }

        std::vector<char> ptx;
        if (ptxSize > 0 && prog) {
            ptx.resize(ptxSize);
            NVRTC_SAFE_CALL(hiprtcGetCode(prog, ptx.data()));
        }

        if (prog) {
            NVRTC_SAFE_CALL(hiprtcDestroyProgram(&prog));
            prog = nullptr;
        }

//        if (module) {
//            CU_SAFE_CALL(hipModuleUnload(module));
//            module = nullptr;
//        }

        if (!ptx.empty()) {
            CU_SAFE_CALL(hipModuleLoadDataEx(&module, ptx.data(), 0, 0, 0));
        }

        if (module && !lowered_kernel_name_float_str.empty()) {
            CU_SAFE_CALL(hipModuleGetFunction(&kernelFloat, module, lowered_kernel_name_float_str.c_str()));
        } else if (!module) { /* Module not loaded */ } else { /* Name string empty, handle as error */ }


        if (module && !lowered_kernel_name_double_str.empty()) {
            CU_SAFE_CALL(hipModuleGetFunction(&kernelDouble, module, lowered_kernel_name_double_str.c_str()));
        } else if (!module) { /* Module not loaded */ } else { /* Name string empty, handle as error */ }


        if (module && !lowered_kernel_name_ssaa_str.empty()) {
            CU_SAFE_CALL(hipModuleGetFunction(&kernelAntialiasing, module, lowered_kernel_name_ssaa_str.c_str()));
        } else if (!module) { /* Module not loaded */ } else { /* Name string empty, handle as error */ }


    } catch (const std::exception& e) {
        std::cerr << "Exception in NVRTC setup/compilation/loading: " << e.what() << std::endl;
        if (prog) {
            NVRTC_SAFE_CALL(hiprtcDestroyProgram(&prog));
        }
        return;
    }
    std::cout << "Kernel loaded successfully.\n";
    custom_formula = true;
}

template <typename Derived>
/// \brief Set the context for the fractal rendering.
/// This function allows switching between CUDA and NVRTC contexts.
/// It frees existing resources and allocates new ones based on the selected context.
void FractalBase<Derived>::set_context(context_type contx) {
    if(contx == context) return;

    // Free existing resources
    FREE_ALL_IMAGE_MEMORY();
    FREE_ALL_NON_IMAGE_MEMORY();

    if(context == context_type::CUDA) {
        if(!initialized_nvrtc){
            CU_SAFE_CALL(hipInit(0));
        }
        initialized_nvrtc = true;
        CU_SAFE_CALL(hipDeviceGet(&device, 0));
        CU_SAFE_CALL(hipCtxCreate(&ctx, 0, device));
        CU_SAFE_CALL(hipCtxSetCurrent(ctx));
        created_context = true;
    }
    else {
        if (ctx) {
            created_context = false;
            CU_SAFE_CALL(hipCtxDestroy(ctx));
            ctx = nullptr;
        }
        if (device) {
            device = 0;
        }
        CUDA_SAFE_CALL(hipSetDevice(0));
    }
    context = contx;

    ALLOCATE_ALL_IMAGE_MEMORY();
    ALLOCATE_ALL_NON_IMAGE_MEMORY();

    if(context == context_type::NVRTC) custom_formula = true;
    else custom_formula = false;
}

template <typename Derived>
context_type FractalBase<Derived>::get_context() { return context; }



template <typename Derived>
bool FractalBase<Derived>::get_bool_custom_formula() { return custom_formula; }

template <>
void FractalBase<fractals::mandelbrot>::render(render_state quality) {
    if (!isCudaAvailable) {
        // --- CPU Rendering Fallback ---
        // This block executes only if CUDA (GPU acceleration) is not available.

        if (quality == render_state::best) {
            return;
        }

        // --- Global CPU Rendering Lock ---
        // Use an atomic flag to prevent multiple CPU rendering processes from running concurrently.
        bool expected_state = false; // We expect no CPU render to be currently running.
        // Attempt to atomically set g_isCpuRendering to true IF it's currently false.
        // memory_order_acq_rel: Ensures memory operations before this in other threads are visible (acquire)
        //                      and memory operations after this in this thread are visible to others (release).
        //                      Crucial for coordinating access to shared rendering resources.
        if (!g_isCpuRendering.compare_exchange_strong(expected_state, true, std::memory_order_acq_rel)) {
            // If the exchange failed, it means g_isCpuRendering was already true. Another CPU render is active.
            return;
        }

        // --- Launch Background CPU Rendering Thread ---
        // Offload the CPU rendering to a separate thread to avoid blocking the main application thread.
        std::thread main_thread([&]() { // Lambda captures necessary variables by reference.

            // --- Automatic Lock Release ---
            // This RAII guard ensures g_isCpuRendering is set back to false when the lambda finishes
            // execution, even if an exception occurs. This releases the global rendering lock.
            RenderGuard renderGuard(g_isCpuRendering);

            // --- Prepare for New Render Task & Divide Work ---
            unsigned int max_threads_local = std::thread::hardware_concurrency();
            if (max_threads_local == 0) max_threads_local = 1;
            render_targets.clear(); // Clear previous task definitions.

            // Distribute image rows among available threads, handling remainders.
            unsigned int rows_per_thread = basic_height / max_threads_local;
            unsigned int remainder_rows = basic_height % max_threads_local;
            unsigned int current_y_start = 0;
            for (unsigned int i = 0; i < max_threads_local; ++i) {
                unsigned int rows_for_this_thread = rows_per_thread + (i < remainder_rows ? 1 : 0);
                if (rows_for_this_thread == 0 && current_y_start >= basic_height) continue; // Avoid empty tasks
                unsigned int y_start = current_y_start;
                unsigned int y_end = current_y_start + rows_for_this_thread;
                if (y_start < y_end) { // Ensure valid range
                    render_targets.emplace_back(0, y_start, basic_width, y_end);
                    current_y_start = y_end;
                } else if (current_y_start >= basic_height) {
                    break; // Stop if all rows are assigned
                }
            }

            unsigned int actual_threads_to_launch = render_targets.size();

            // --- Thread Limit Check ---
            // Ensure we don't try to launch more threads than we have control flags for.
            if (actual_threads_to_launch > thread_stop_flags.size()) {
                std::cerr << "Warning: Clamping required threads (" << actual_threads_to_launch
                          << ") to available flags (" << thread_stop_flags.size() << ")." << std::endl;
                actual_threads_to_launch = thread_stop_flags.size();
                // This could result in parts of the image not being rendered.
            }

            if (actual_threads_to_launch == 0 && basic_height > 0 && basic_width > 0) {
                // Edge case: image has dimensions, but no threads assigned (e.g., height too small or clamped).
                return; // Exit lambda.
            }

            // --- Launch Worker Threads ---
            for (unsigned int i = 0; i < actual_threads_to_launch; ++i) {
                // Set flag state to '0' (working).
                // memory_order_release: Make this write visible before the thread potentially reads it.
                thread_stop_flags[i].store(0, std::memory_order_release);

                // Launch the actual rendering function (e.g., cpu_render_mandelbrot) in a new thread.
                std::thread t(cpu_render_mandelbrot, render_targets[i], pixels, basic_width, basic_height,
                              zoom_x, zoom_y, x_offset, y_offset, palette.data(), paletteSize,
                              max_iterations, h_total_iterations, std::ref(thread_stop_flags[i])); // Pass flag by ref

                // Detach the worker thread: The main_thread won't wait (join) for it directly.
                // Completion is tracked using the atomic stop flags.
                t.detach();
            }

            // --- Wait for Completion & Intermediate Updates ---
            // This loop polls the status flags of the worker threads.
            while (true) {
                bool all_done = true;
                if (actual_threads_to_launch > 0) {
                    for(unsigned int i = 0; i < actual_threads_to_launch; ++i) {
                        // Check if the flag is '1' (finished).
                        // memory_order_acquire: Ensures reads here see the final writes from the worker.
                        if (thread_stop_flags[i].load(std::memory_order_acquire) != 1) {
                            all_done = false;
                            break;
                        }
                    }
                }

                if (all_done) {
                    break; // All workers finished.
                }

                // Allows updating the display or performing other tasks periodically while rendering.
                post_processing();
                // Sleep briefly to avoid pegging the CPU core running this management thread.
                std::this_thread::sleep_for(std::chrono::microseconds(1));
            }

            // Final update after all threads are confirmed done.
            post_processing();

        }); // End of lambda for main_thread

        // Detach the management thread itself. This makes the entire CPU render operation
        // asynchronous from the perspective of the function caller. The OS handles the detached thread.
        // The RenderGuard inside the lambda ensures the global lock is eventually released.
        main_thread.detach();
        return;

    } // End of if (!isCudaAvailable)
    unsigned int old_width = width, old_height = height;

    double new_zoom_scale;

    if (quality == render_state::good) {
        width = basic_width;
        height = basic_height;
        antialiasing = false;
        new_zoom_scale = 1.0;
    }
    else { // render_state::best, Antialiasing -> more pixels need to be rendered
        width = basic_width * 2;
        height = basic_height * 2;
        antialiasing = true;
        new_zoom_scale = 2.0;
    }

    if (width != old_width || height != old_height) {
        double center_x = x_offset + (old_width / (zoom_x * zoom_scale)) / 2.0;
        double center_y = y_offset + (old_height / (zoom_y * zoom_scale)) / 2.0;

        zoom_scale = new_zoom_scale;

        x_offset = center_x - (width / (zoom_x * zoom_scale)) / 2.0;
        y_offset = center_y - (height / (zoom_y * zoom_scale)) / 2.0;
    }

    double render_zoom_x = zoom_x * zoom_scale;
    double render_zoom_y = zoom_y * zoom_scale;
    size_t len = width * height * 4;
    if(!custom_formula){
        if (render_zoom_x > 1e7) {
            dimBlock = dim3(10, 10);
            dimGrid = dim3(
                    (width + dimBlock.x - 1) / dimBlock.x,
                    (height + dimBlock.y - 1) / dimBlock.y
            );
            fractal_rendering<double><<<dimGrid, dimBlock, 0, stream>>>(
                    d_pixels, len, width, height, render_zoom_x, render_zoom_y,
                    x_offset, y_offset, d_palette, paletteSize,
                    max_iterations, d_total_iterations
            );
        }
        else {
            dimBlock = dim3(32, 32);
            dimGrid = dim3(
                    (width + dimBlock.x - 1) / dimBlock.x,
                    (height + dimBlock.y - 1) / dimBlock.y
            );

            fractal_rendering<float><<<dimGrid, dimBlock, 0, stream>>>(
                    d_pixels, len, width, height, render_zoom_x, render_zoom_y,
                    x_offset, y_offset, d_palette, paletteSize,
                    max_iterations, d_total_iterations
            );
        }
    }
    else { // Custom, Formula handling
        hipCtxSetCurrent(ctx);
        dimBlock = dim3(32, 32);
        dimGrid = dim3(
                (width + dimBlock.x - 1) / dimBlock.x,
                (height + dimBlock.y - 1) / dimBlock.y
        );

        size_t len = width * height * 4;
        double render_zoom_x_d = render_zoom_x;
        double render_zoom_y_d = render_zoom_y;
        double x_offset_d = x_offset;
        double y_offset_d = y_offset;
        float max_iterations_val = max_iterations;
        double max_iterations_d = max_iterations;
        unsigned int paletteSize_val = paletteSize;
        unsigned int width_val = width;
        unsigned int height_val = height;

        if (zoom_x > 1e7) {
            void* args[] = {
                    &cu_d_pixels,
                    &len,
                    &width_val,
                    &height_val,
                    &render_zoom_x_d,
                    &render_zoom_y_d,
                    &x_offset_d,
                    &y_offset_d,
                    &cu_palette,
                    &paletteSize_val,
                    &max_iterations_d,
                    &cu_d_total_iterations
            };
            CU_SAFE_CALL(hipModuleLaunchKernel(kernelDouble, dimGrid.x, dimGrid.y, 1,
                                        dimBlock.x, dimBlock.y, 1,
                                        0, nullptr,
                                        args, nullptr));
        } else { // Запуск float-версии
            float render_zoom_x_f = static_cast<float>(render_zoom_x_d);
            float render_zoom_y_f = static_cast<float>(render_zoom_y_d);
            float x_offset_f = static_cast<float>(x_offset_d);
            float y_offset_f = static_cast<float>(y_offset_d);

            void* args[] = {
                    &cu_d_pixels,
                    &len,
                    &width_val,
                    &height_val,
                    &render_zoom_x_f,
                    &render_zoom_y_f,
                    &x_offset_f,
                    &y_offset_f,
                    &cu_palette,
                    &paletteSize_val,
                    &max_iterations_val,
                    &cu_d_total_iterations
            };
            CU_SAFE_CALL(hipModuleLaunchKernel(kernelFloat,
                                        dimGrid.x, dimGrid.y, 1,
                                        dimBlock.x, dimBlock.y, 1,
                                        0, nullptr,
                                        args, nullptr));
        }

    }


    hipError_t err = hipGetLastError();
    while (err != hipSuccess && context != context_type::NVRTC) {
        dimBlock.x -= 2;
        dimBlock.y -= 2;
        dimGrid.x = (width + dimBlock.x - 1) / dimBlock.x;
        dimGrid.y = (height + dimBlock.y - 1) / dimBlock.y;
        if (zoom_x > 1e7) {
            fractal_rendering<double><<<dimGrid, dimBlock, 0, stream>>>(
                d_pixels, len, width, height, render_zoom_x, render_zoom_y,
                x_offset, y_offset, d_palette, paletteSize,
                max_iterations, d_total_iterations
                );
        }
        else {
            fractal_rendering<float><<<dimGrid, dimBlock, 0, stream>>>(
                    d_pixels, len, width, height, render_zoom_x, render_zoom_y,
                    x_offset, y_offset, d_palette, paletteSize,
                    max_iterations, d_total_iterations
            );
        }
        err = hipGetLastError();
        if (dimBlock.x < 3) {
            std::cerr << "Critical Issue (mandelbrot set): " << hipGetErrorString(err) << "\n";
            return;
        }
    }
    ++counter;
    post_processing();
}

template <>
void FractalBase<fractals::julia>::render(
    render_state quality,
    double zx, double zy
) {
    if (!isCudaAvailable) {
        return;
    }

    unsigned int old_width = width, old_height = height;

    double new_zoom_scale;

    if (quality == render_state::good) {
        width = basic_width;
        height = basic_height;
        antialiasing = false;
        new_zoom_scale = 1.0;
    }
    else { // render_state::best, Antialiasing -> more pixels need to be rendered
        width = basic_width * 2;
        height = basic_height * 2;
        antialiasing = true;
        new_zoom_scale = 2.0;
    }

    if (width != old_width || height != old_height) {
        double center_x = x_offset + (old_width / (zoom_x * zoom_scale)) / 2.0;
        double center_y = y_offset + (old_height / (zoom_y * zoom_scale)) / 2.0;

        zoom_scale = new_zoom_scale;

        x_offset = center_x - (width / (zoom_x * zoom_scale)) / 2.0;
        y_offset = center_y - (height / (zoom_y * zoom_scale)) / 2.0;
    }

    double render_zoom_x = zoom_x * zoom_scale;
    double render_zoom_y = zoom_y * zoom_scale;

    dim3 dimBlock(32, 32);
    dim3 dimGrid(
        (width + dimBlock.x - 1) / dimBlock.x,
        (height + dimBlock.y - 1) / dimBlock.y
    );


    size_t len = width * height * 4;

    if(!custom_formula){
        if (render_zoom_x > 1e7) {
            dimBlock = dim3(10, 10);
            dimGrid = dim3(
                    (width + dimBlock.x - 1) / dimBlock.x,
                    (height + dimBlock.y - 1) / dimBlock.y
            );
            fractal_rendering<double><<<dimGrid, dimBlock, 0, stream>>>(
                    d_pixels, len, width, height, render_zoom_x, render_zoom_y,
                    x_offset, y_offset, d_palette, paletteSize,
                    max_iterations, d_total_iterations, zx, zy
            );
        }
        else {
            dimBlock = dim3(32, 32);
            dimGrid = dim3(
                    (width + dimBlock.x - 1) / dimBlock.x,
                    (height + dimBlock.y - 1) / dimBlock.y
            );
            fractal_rendering<float><<<dimGrid, dimBlock, 0, stream>>>(
                    d_pixels, len, width, height, render_zoom_x, render_zoom_y,
                    x_offset, y_offset, d_palette, paletteSize,
                    max_iterations, d_total_iterations, zx, zy
            );
        }
    }
    else {
        hipCtxSetCurrent(ctx);
        double maxI = max_iterations;
        float maxI_f = maxI;

        void* args[] = {
                &cu_d_pixels,
                &len,
                &width,
                &height,
                &render_zoom_x,
                &render_zoom_y,
                &x_offset,
                &y_offset,
                &cu_palette,
                &paletteSize,
                &maxI,
                &cu_d_total_iterations,
                &zx,
                &zy
        };

        hipFunction_t launch_kernel = zoom_x > 1e7 ? kernelDouble : kernelFloat;
        CU_SAFE_CALL(hipModuleLaunchKernel(launch_kernel,
                dimGrid.x, dimGrid.y, 1,
                dimBlock.x, dimBlock.y, 1,
                0, nullptr,
                args, nullptr));
    }

    hipError_t err = hipGetLastError();
    while (err != hipSuccess && context != context_type::NVRTC) {
        dimBlock.x -= 2;
        dimBlock.y -= 2;
        dimGrid.x = (width + dimBlock.x - 1) / dimBlock.x;
        dimGrid.y = (height + dimBlock.y - 1) / dimBlock.y;
        if (zoom_x > 1e7) {
            fractal_rendering<double><<<dimGrid, dimBlock, 0, stream>>>(
                d_pixels, len, width, height, render_zoom_x, render_zoom_y,
                x_offset, y_offset, d_palette, paletteSize,
                max_iterations, d_total_iterations, zx, zy
            );
        }
        else {
            fractal_rendering<float><<<dimGrid, dimBlock, 0, stream>>>(
                    d_pixels, len, width, height, float(render_zoom_x), float(render_zoom_y),
                    x_offset, y_offset, d_palette, paletteSize,
                    max_iterations, d_total_iterations, zx, zy
            );
        }
        err = hipGetLastError();
        if (dimBlock.x < 2) {
            std::cout << "Critical Issue (julia set): " << hipGetErrorString(err) << "\n";
        }
    }
    ++counter;
    post_processing();
}

template <>
void FractalBase<fractals::mandelbrot>::drawIterationLines(sf::Vector2i mouse_pos) {
    int curr_iter = 0;
    double zr = 0.0;
    double zi = 0.0;

    if (max_iterations >= iterationpoints.size()) {
        iterationpoints.resize(max_iterations * 2);
    }

    double cr = mouse_pos.x / zoom_x - x_offset;
    double ci = mouse_pos.y / zoom_y - y_offset;

    while (curr_iter < max_iterations && zr * zr + zi * zi < 4.0) {
        double tmp_zr = zr;
        zr = zr * zr - zi * zi + cr;
        zi = 2.0 * tmp_zr * zi + ci;

        double x = (zr + x_offset) * zoom_x;
        double y = (zi + y_offset) * zoom_y;

        iterationpoints[curr_iter].position = sf::Vector2f(x, y);
        iterationpoints[curr_iter].color = sf::Color::Red;
        curr_iter++;
    }

    iterationline.create(curr_iter);
    iterationline.update(iterationpoints.data());
    drawen_iterations = curr_iter;
}

template <>
void FractalBase<fractals::julia>::drawIterationLines(sf::Vector2i mouse_pos) {
    int curr_iter = 0;
    double zr = 0.0;
    double zi = 0.0;

    // Calculate cr and ci based on the mouse position and current view parameters
    double cr = x_offset + (mouse_pos.x / zoom_x);
    double ci = y_offset + (mouse_pos.y / zoom_y);

    while (curr_iter < max_iterations && zr * zr + zi * zi < 4.0) {
        double tmp_zr = zr;
        zr = zr * zr - zi * zi + cr;
        zi = 2.0 * tmp_zr * zi + ci;

        // Map the complex number z back to screen coordinates
        float x = static_cast<float>((zr - x_offset) * zoom_x);
        float y = static_cast<float>((zi - y_offset) * zoom_y);

        if (curr_iter < iterationpoints.size()) {
            iterationpoints[curr_iter].position = sf::Vector2f(x, y);
            iterationpoints[curr_iter].color = sf::Color::Red;
        }

        curr_iter++;
    }

    iterationline.create(curr_iter);
    iterationline.update(iterationpoints.data());
    drawen_iterations = curr_iter;
}


template <typename Derived>
void FractalBase<Derived>::draw(sf::RenderTarget& target, sf::RenderStates states) const {
     states.transform *= getTransform();
     target.draw(sprite, states);
     target.draw(iterationline, 0, drawen_iterations, states);
}
template <typename Derived>

// Mouse pos should be relative to the picture and not to the screen
void FractalBase<Derived>::handleZoom(double wheel_delta, const sf::Vector2i mouse_pos) {

    double old_zoom_x = zoom_x;
    double old_zoom_y = zoom_y;
    double old_x_offset = x_offset;
    double old_y_offset = y_offset;

    double zoom_change = 1.0 + wheel_delta * zoom_speed;
    zoom_factor *= zoom_change;
    zoom_factor = std::max(std::min(zoom_factor, 100000000000000.0), 0.01);

    zoom_x = basic_zoom_x * zoom_factor;
    zoom_y = basic_zoom_y * zoom_factor;

    double image_mouse_x = mouse_pos.x * 1.0;
    double image_mouse_y = mouse_pos.y * 1.0;


    x_offset = old_x_offset + (image_mouse_x / zoom_x - image_mouse_x / old_zoom_x);
    y_offset = old_y_offset + (image_mouse_y / zoom_y - image_mouse_y / old_zoom_y);

}

template <typename Derived>
void FractalBase<Derived>::start_dragging(sf::Vector2i mouse_pos) {
    is_dragging = true;
    drag_start_pos = mouse_pos;
}


template <typename Derived>
void FractalBase<Derived>::dragging(sf::Vector2i mouse_pos) {
    if (!is_dragging) return;

    sf::Vector2i delta_pos = mouse_pos - drag_start_pos;
    double delta_real = static_cast<double>(delta_pos.x) / (zoom_x * zoom_scale);
    double delta_imag = static_cast<double>(delta_pos.y) / (zoom_y * zoom_scale);

    x_offset += delta_real;
    y_offset += delta_imag;
    drag_start_pos = mouse_pos;
}

template <typename Derived>
void FractalBase<Derived>::stop_dragging() {
    is_dragging = false;
}

template <typename Derived>
void FractalBase<Derived>::move_fractal(sf::Vector2i offset) {
    x_offset += offset.x / (zoom_x * zoom_scale);
	y_offset += offset.y / (zoom_y * zoom_scale);
}

template <>
void FractalBase<fractals::julia>::start_timelapse() {
    timelapse.zx = disX(gen);
    timelapse.zy = disY(gen);
    timelapse.velocityX = disVelX(gen);
	timelapse.velocityY = disVelY(gen);
}

template <>
void FractalBase<fractals::julia>::update_timelapse() {
    const auto elapsed = clock.getElapsedTime();
    const auto elapsedMs = elapsed.asMilliseconds();
    constexpr float frameTimeMs = 1000.0f / 360.0f;

    if (elapsedMs <= frameTimeMs) return;

    float deltaTime = elapsedMs / 1000.0f;
    clock.restart();

    constexpr float GRAVITATIONAL_STRENGTH = 0.1f;
    constexpr float EVENT_HORIZON = 0.1f;
    constexpr float MAX_VEL = 20.0f;
    constexpr float VELOCITY_DAMPING = 0.9995f;
    constexpr float TARGET_CHANGE_TIME = 2.0f;
    constexpr float TRANSITION_SPEED = 3.5f;

    static float timeToTargetChange = TARGET_CHANGE_TIME;
    static float targetZX = 0.0f;
    static float targetZY = 0.0f;
    static float currentTargetZX = 0.0f;
    static float currentTargetZY = 0.0f;

    timeToTargetChange -= deltaTime;
    if (timeToTargetChange <= 0) {
        timeToTargetChange = TARGET_CHANGE_TIME;
        targetZX = (static_cast<float>(rand()) / RAND_MAX) * 1.0f - 0.5f;
        targetZY = (static_cast<float>(rand()) / RAND_MAX) * 1.0f - 0.5f;
    }

    currentTargetZX += (targetZX - currentTargetZX) * TRANSITION_SPEED * deltaTime;
    currentTargetZY += (targetZY - currentTargetZY) * TRANSITION_SPEED * deltaTime;

    float dx = currentTargetZX - timelapse.zx;
    float dy = currentTargetZY - timelapse.zy;
    float distanceSq = dx * dx + dy * dy + 0.0001f;

    timelapse.velocityX += GRAVITATIONAL_STRENGTH * dx * deltaTime;
    timelapse.velocityY += GRAVITATIONAL_STRENGTH * dy * deltaTime;

    float distance = sqrt(distanceSq);
    if (distance < EVENT_HORIZON) {
        float angle = atan2(dy, dx);
        float boost = (1.0f - (distance / EVENT_HORIZON)) * 4.0f;
        timelapse.velocityX += cos(angle + 3.14159265359f / 2.0f) * boost * deltaTime;
        timelapse.velocityY += sin(angle + 3.14159265359f / 2.0f) * boost * deltaTime;
    }

    timelapse.velocityX *= VELOCITY_DAMPING;
    timelapse.velocityY *= VELOCITY_DAMPING;
    timelapse.velocityX = std::clamp(timelapse.velocityX, -MAX_VEL, MAX_VEL);
    timelapse.velocityY = std::clamp(timelapse.velocityY, -MAX_VEL, MAX_VEL);

    timelapse.zx += timelapse.velocityX * deltaTime;
    timelapse.zy += timelapse.velocityY * deltaTime;

    constexpr float BOUNDARY = 2.0f;
    if (std::abs(timelapse.zx) > BOUNDARY) {
        timelapse.zx = std::copysign(BOUNDARY, timelapse.zx);
        timelapse.velocityX *= -0.5f;
    }
    if (std::abs(timelapse.zy) > BOUNDARY) {
        timelapse.zy = std::copysign(BOUNDARY, timelapse.zy);
        timelapse.velocityY *= -0.5f;
    }

    render(render_state::good, timelapse.zx, timelapse.zy);
}

template <>
void FractalBase<fractals::julia>::stop_timelapse() {
	timelapse.zx = 0;
	timelapse.zy = 0;
	timelapse.velocityX = 0;
	timelapse.velocityY = 0;
}

template class FractalBase<fractals::mandelbrot>;
template class FractalBase<fractals::julia>;