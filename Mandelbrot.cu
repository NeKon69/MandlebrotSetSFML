#include "hip/hip_runtime.h"
#include <SFML/Graphics.hpp>
#include <hip/hip_runtime.h>
#include <complex>
#include <cmath>
#include <vector_alias.hpp>
#include <unordered_map>
#include <random>
#include <algorithm>
#include <thread>


/**
 * @brief Maximum number of iterations for Mandelbrot set calculation.
 * Higher values increase detail but also computation time.
 */
unsigned int max_iterations = 2000;

double basic_zoom_x = 240.0f;
double basic_zoom_y = 240.0f;

double zoom_x = basic_zoom_x;
double zoom_y = basic_zoom_y;

double x_offset = 3.5f;
double y_offset = 2.5f;

double zoom_factor = 1.0f;
double zoom_speed = 0.1f;

sf::Vector2i drag_start_pos;
bool is_dragging = false;

unsigned int width = 1600;
unsigned int height = 1200;

/**
 * @brief Pixel buffer in host memory to store the rendered image data.
 */
unsigned char* pixels = new unsigned char[width * height * 4];

/**
 * @brief Pixel buffer in device (GPU) memory, used for CUDA kernel.
 */
unsigned char* d_pixels;

#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

__device__ double complex_mult_real(double real1, double imag1, double real2, double imag2) {
    return real1 * real2 - imag1 * imag2;
}

__device__ double complex_mult_imag(double real1, double imag1, double real2, double imag2) {
    return real1 * imag2 + imag1 * real2;
}

__device__ double dev_abs(double x) {
    return (x >= 0) ? x : -x;
}

__device__ double dev_log2(double x) {
    return log(x) / log(2.0);
}

__device__ double dev_sqrt(double x) {
    return sqrt(x);
}

__device__ double complex_abs2(double real, double imag) {
    return real * real + imag * imag;
}

__device__ double Gradient(double current_iteration, double max_iter) {
    if (current_iteration >= max_iter) return 0.0;
    current_iteration = dev_sqrt(current_iteration);
    max_iter = dev_sqrt(max_iter);
    return (current_iteration) / static_cast<double>(max_iter);
}

__device__ sf::Color getPaletteColor(int index, int paletteSize, sf::Color* palette) {
    index = (index < 0) ? 0 : ((index >= paletteSize) ? paletteSize - 1 : index);
    return palette[index];
}


void HSVtoRGB(double h, double s, double v, unsigned int& r, unsigned int& g, unsigned int& b) {
    h = fmod(h, 360.0);
    double c = v * s;
    double x = c * (1 - std::fabs(fmod(h / 60.0, 2) - 1));
    double m = v - c;

    double r_, g_, b_;
    if (h < 60) { r_ = c, g_ = x, b_ = 0; }
    else if (h < 120) { r_ = x, g_ = c, b_ = 0; }
    else if (h < 180) { r_ = 0, g_ = c, b_ = x; }
    else if (h < 240) { r_ = 0, g_ = x, b_ = c; }
    else if (h < 300) { r_ = x, g_ = 0, b_ = c; }
    else { r_ = c, g_ = 0, b_ = x; }

    r = static_cast<unsigned int>((r_ + m) * 255);
    g = static_cast<unsigned int>((g_ + m) * 255);
    b = static_cast<unsigned int>((b_ + m) * 255);
}


/**
 * @brief Creates a color palette using HSV color space and converts it to RGB.
 * The palette is designed to smoothly transition colors for visualization of the Mandelbrot set.
 * @param numColors The number of colors in the palette.
 * @return std::vector<sf::Color> A vector containing the generated color palette.
 */
std::vector<sf::Color> createHSVPalette(int numColors) {
    std::vector<sf::Color> palette;
    for (int i = 0; i < numColors; ++i) {
        double t = static_cast<double>(i) / numColors;

        double hue;
        double saturation = 1.0;
        double value;

        if (t < 0.2) {
            hue = 240.0;
            value = 0.3 + t * (1.0 - 0.3) / 0.2;
        }
        else {
            hue = std::pow((t - 0.2) / 0.8, 0.5) * 360.0;
            value = 1.0;
        }

        unsigned int r, g, b;
        HSVtoRGB(hue, saturation, value, r, g, b);
        palette.push_back(sf::Color(r, g, b, 255));
    }
    return palette;
}

std::vector<sf::Color> palette = createHSVPalette(200000);
sf::Image image({ 1600, 1200 }, sf::Color::Black);


/**
 * @brief CUDA kernel function to calculate and render the Mandelbrot set.
 * This kernel is executed in parallel by multiple threads on the GPU.
 * Each thread calculates the color of a single pixel based on its position
 * and the Mandelbrot set algorithm.
 *
 * @param pixels Pointer to the pixel data buffer in device memory.
 * @param width Image width.
 * @param height Image height.
 * @param zoom_x Zoom level along the x-axis.
 * @param zoom_y Zoom level along the y-axis.
 * @param x_offset Offset in the x-direction to move the view.
 * @param y_offset Offset in the y-direction to move the view.
 * @param d_palette Color palette in device memory to color the Mandelbrot set.
 * @param paletteSize Size of the color palette.
 * @param maxIterations Maximum iterations for Mandelbrot calculation.
 */
__global__ void mandelbrot(unsigned char* pixels, int width, int height, double zoom_x, double zoom_y, double x_offset, double y_offset, sf::Color* d_palette, int paletteSize, double maxIterations) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        double real = x / zoom_x - x_offset;
        double imag = y / zoom_y - y_offset;
        double z_real = 0.0;
        double z_imag = 0.0;
        double current_iteration = 0;

        while (complex_abs2(z_real, z_imag) < 4 && current_iteration < maxIterations) {
            double new_real = complex_mult_real(z_real, z_imag, z_real, z_imag) + real;
            double new_imag = complex_mult_imag(z_real, z_imag, z_real, z_imag) + imag;

            // Some strange thing
            //double new_real = complex_mult_real(z_real, z_imag, z_real, z_imag) * new_real + real;
            //double new_imag = complex_mult_imag(z_real, z_imag, z_real, z_imag) * new_imag + imag;
            z_real = new_real;
            z_imag = new_imag;
            current_iteration++;
        }

        unsigned char r, g, b;
        if (current_iteration == maxIterations) {
            r = g = b = 0;
        }
        else {
            // Color gradient calculation based on iteration count for points outside the set
            double max_iters = maxIterations;

            // Smooth iteration count
            current_iteration = current_iteration + 1 - dev_log2(dev_log2(dev_abs(dev_sqrt(complex_abs2(z_real, z_imag)))));

            // Calculate gradient value
            double gradient = Gradient(current_iteration, max_iters);

            // Map gradient to palette index
            int index = static_cast<int>(gradient * (paletteSize - 1));
            sf::Color color = getPaletteColor(index, paletteSize, d_palette);
            r = color.r;
            g = color.g;
            b = color.b;
        }
        int index = (y * width + x) * 4;
        pixels[index + 0] = r;
        pixels[index + 1] = g;
        pixels[index + 2] = b;
        pixels[index + 3] = 255;
    }
}



/**
 * @brief Applies 4x Super-Sampling Anti-Aliasing (SSAA) to reduce aliasing artifacts.
 * This function takes a high-resolution image (2x width and height of the desired output)
 * and downsamples it to produce a smoother, anti-aliased image. It averages the color
 * of 2x2 pixel blocks in the high-resolution image to generate each pixel in the
 * low-resolution image.
 *
 * @param high_qual The high-resolution input image.
 * @param size The scaling factor (should be 2 for 4x SSAA).
 * @return sf::Image The anti-aliased, low-resolution image.
 */
static sf::Image ANTIALIASING_SSAA4(sf::Image& high_qual, unsigned int size) {
    sf::Image low_qual({ high_qual.getSize().x / size * 2, high_qual.getSize().y / size * 2 }, sf::Color::Black);
    for (unsigned int x = 0; x < high_qual.getSize().x; x += 2) {
        for (unsigned int y = 0; y < high_qual.getSize().y; y += 2) {
            sf::Color colorTopLeft = high_qual.getPixel({ x, y });
            sf::Color colorTopRight = high_qual.getPixel({ x + 1, y });
            sf::Color colorBottomLeft = high_qual.getPixel({ x, y + 1 });
            sf::Color colorBottomRight = high_qual.getPixel({ x + 1, y + 1 });
            sf::Color color = sf::Color((colorTopLeft.r + colorTopRight.r + colorBottomLeft.r + colorBottomRight.r) / 4, (colorTopLeft.g + colorTopRight.g + colorBottomLeft.g + colorBottomRight.g) / 4, (colorTopLeft.b + colorTopRight.b + colorBottomLeft.b + colorBottomRight.b) / 4);
            low_qual.setPixel({ x / 2, y / 2 }, color);
        }
    }
    return low_qual;
}


/**
 * @brief Handles zoom functionality based on mouse wheel input.
 * Adjusts the zoom factor and offsets to zoom in or out centered around the mouse position.
 *
 * @param wheel_delta The direction and magnitude of the mouse wheel scroll.
 * @param mouse_pos The current mouse position in window coordinates.
 */
void handleZoom(float wheel_delta, const sf::Vector2i mouse_pos) {
    double old_zoom_x = zoom_x;
    double old_zoom_y = zoom_y;
    double old_x_offset = x_offset;
    double old_y_offset = y_offset;

    double zoom_change = 1.0 + wheel_delta * zoom_speed;
    zoom_factor *= zoom_change; // Update zoom factor based on wheel delta

    zoom_factor = std::max(std::min(zoom_factor, 1000000000000.0), 0.01);

    zoom_x = basic_zoom_x * zoom_factor;
    zoom_y = basic_zoom_y * zoom_factor;

    // Calculate the change in offset to keep the zoom centered at the mouse position
    double image_mouse_x = mouse_pos.x * (1600.0 / 800.0);
    double image_mouse_y = mouse_pos.y * (1200.0 / 600.0);

    x_offset = old_x_offset + (image_mouse_x / zoom_x - image_mouse_x / old_zoom_x);
    y_offset = old_y_offset + (image_mouse_y / zoom_y - image_mouse_y / old_zoom_y);

    std::cout << "Zoom Factor: " << zoom_factor << ", Offset: (" << x_offset << ", " << y_offset << ")" << std::endl;
}


/**
 * @brief Starts the dragging operation when the mouse button is pressed.
 * Records the starting mouse position.
 *
 * @param mouse_pos The current mouse position.
 */
void start_dragging(sf::Vector2i mouse_pos) {
    is_dragging = true;
    drag_start_pos = mouse_pos;
}


/**
 * @brief Handles the dragging operation while the mouse is moved and the button is held down.
 * Updates the view offset based on the mouse movement since the drag started.
 *
 * @param mouse_pos The current mouse position.
 */
void dragging(sf::Vector2i mouse_pos) {
    if (!is_dragging) return;

    sf::Vector2i delta_pos = mouse_pos - drag_start_pos;

    double delta_real = static_cast<double>(delta_pos.x) / zoom_x;
    double delta_imag = static_cast<double>(delta_pos.y) / zoom_y;

    x_offset += delta_real;
    y_offset += delta_imag;

    drag_start_pos = mouse_pos;
}



/**
 * @brief Stops the dragging operation when the mouse button is released.
 * Resets the dragging flag.
 */
void stop_dragging() {
    is_dragging = false;
}



int main() {
	// Used to render or not the image based on the moves made with the mouse
    bool need_render = true;
    sf::RenderWindow window(sf::VideoMode({ 800, 600 }), "Mandelbrot");
    sf::Image image({ 1600, 1200 }, sf::Color::Black);
    sf::Image compressed({ 800, 600 }, sf::Color::Black);
    sf::Clock timer;

    double max_iters = max_iterations;

    sf::Vector2i mouse;

    // Palette transfer to device memory for CUDA kernel access
    sf::Color* d_palette;
    int paletteSize = palette.size();
    hipMalloc(&d_palette, paletteSize * sizeof(sf::Color)); // Allocate device memory for palette
    hipMemcpy(d_palette, palette.data(), paletteSize * sizeof(sf::Color), hipMemcpyHostToDevice); // Copy palette data to device

    hipMalloc(&d_pixels, width * height * 4 * sizeof(unsigned char)); // Allocate device memory for pixels

    while (window.isOpen()) {
        while (const auto event = window.pollEvent()) {
            if (const auto* mm = event->getIf<sf::Event::MouseMoved>()) mouse = mm->position;

            if (event->is<sf::Event::Closed>()) {
                window.close();
            }

            if (const auto* button = event->getIf<sf::Event::KeyPressed>()) {
                if (button->scancode == sf::Keyboard::Scancode::Escape) {
                    window.close();
                }
            }

            if (const auto* mouseWheelScrolled = event->getIf<sf::Event::MouseWheelScrolled>()) {
                need_render = true;
                handleZoom(mouseWheelScrolled->delta, mouse);
            }

            if (const auto* mouseButtonPressed = event->getIf<sf::Event::MouseButtonPressed>()) {
                if (mouseButtonPressed->button == sf::Mouse::Button::Left) {
                    start_dragging(mouse);
                }
            }

            if (const auto* mouseButtonReleased = event->getIf<sf::Event::MouseButtonReleased>()) {
                if (mouseButtonReleased->button == sf::Mouse::Button::Left) {
                    stop_dragging();
                }
            }

            if (const auto* mouseButtonReleased = event->getIf<sf::Event::MouseMoved>()) {
                if (is_dragging) need_render = true;
                dragging({ mouse.x, mouse.y });
            }
        }

        if (need_render) {
            // Configure CUDA kernel execution grid and block dimensions
            dim3 dimBlock(256, 256);
            dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (height + dimBlock.y - 1) / dimBlock.y);

            // Launch Mandelbrot CUDA kernel
            mandelbrot <<<dimBlock, dimGrid>>> (d_pixels, width, height, zoom_x, zoom_y, x_offset, y_offset, d_palette, paletteSize, max_iterations);
            hipDeviceSynchronize();


            // Copy rendered pixels back to host
            hipMemcpy(pixels, d_pixels, width * height * 4 * sizeof(unsigned char), hipMemcpyDeviceToHost);

            sf::Image image1({ width, height }, pixels);
            image = image1;
            need_render = false;


            // Apply 4x SSAA
            compressed = ANTIALIASING_SSAA4(image, 4);

            window.clear();

            sf::Texture tt(compressed);
            sf::Sprite sprite(tt);

            sprite.setPosition({ 0, 0 });

            window.draw(sprite, sprite.getTransform());

            auto time = timer.restart();
            std::cout << "Mandelbrot set was drew in: " << time.asMilliseconds() << std::endl;


            // Display the rendered frame
            window.display();
        }
    }

    // Free device memory
    hipFree(d_pixels);
    hipFree(d_palette);

    return 0;
}